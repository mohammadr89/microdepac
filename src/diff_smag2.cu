#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2023 Chiel van Heerwaarden
 * Copyright (c) 2011-2023 Thijs Heus
 * Copyright (c) 2014-2023 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include <cmath>
#include <algorithm>
#include <iostream>

#include "grid.h"
#include "fields.h"
#include "master.h"
#include "boundary.h"
#include "boundary_surface.h"
#include "defines.h"
#include "constants.h"
#include "thermo.h"
#include "tools.h"
#include "stats.h"
#include "monin_obukhov.h"
#include "fast_math.h"

#include "diff_smag2.h"
#include "diff_kernels.cuh"

// Kernel Launcher
#include "cuda_launcher.h"
#include "diff_smag2_kl_kernels.cuh"
#include "diff_kl_kernels.cuh"

/* Calculate the mixing length (mlen) offline, and put on GPU */
#ifdef USECUDA
template<typename TF>
void Diff_smag2<TF>::prepare_device(Boundary<TF>& boundary)
{
    auto& gd = grid.get_grid_data();

    std::vector<TF> mlen(gd.kcells);

    if (boundary.get_switch() == "default")
    {
        for (int k=0; k<gd.kcells; ++k)
            mlen[k] = cs * pow(gd.dx*gd.dy*gd.dz[k], 1./3.);
    }
    else
    {
        const TF n_mason = TF(2);
        for (int k=0; k<gd.kcells; ++k)
            mlen[k] = std::pow(cs * std::pow(gd.dx*gd.dy*gd.dz[k], TF(1./3.)), n_mason);
    }

    mlen_g.allocate(gd.kcells);
    cuda_safe_call(hipMemcpy(mlen_g, mlen.data(), mlen_g.size_in_bytes(), hipMemcpyHostToDevice));
}

template<typename TF>
void Diff_smag2<TF>::clear_device()
{
}
#endif

#ifdef USECUDA
template<typename TF>
void Diff_smag2<TF>::exec_viscosity(Stats<TF>&, Thermo<TF>& thermo)
{
    namespace dk = Diff_kernels_g;
    auto& gd = grid.get_grid_data();

    // Grid layout struct for cuda launcher.
    Grid_layout grid_layout = {
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.kstart, gd.kend,
            gd.istride,
            gd.jstride,
            gd.kstride};

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kcells);
    dim3 blockGPU(blocki, blockj, 1);

    // Contain the full icells and jcells in this grid.
    const int grid2di  = gd.icells/blocki + (gd.icells%blocki > 0);
    const int grid2dj  = gd.jcells/blockj + (gd.jcells%blockj > 0);

    dim3 grid2dGPU (grid2di, grid2dj);
    dim3 block2dGPU(blocki, blockj);

    // Use surface model.
    if (boundary.get_switch() != "default")
    {
        auto& z0m_g   = boundary.get_z0m_g();

        // Get MO gradients velocity:
        auto& dudz_g  = boundary.get_dudz_g();
        auto& dvdz_g  = boundary.get_dvdz_g();

        // Calculate total strain rate
        launch_grid_kernel<Diff_les_kernels::calc_strain2_g<TF, true>>(
            grid_layout,
            fields.sd.at("evisc")->fld_g.view(),
            fields.mp.at("u")->fld_g,
            fields.mp.at("v")->fld_g,
            fields.mp.at("w")->fld_g,
            dudz_g, dvdz_g,
            gd.dzi_g, gd.dzhi_g,
            gd.dxi, gd.dyi);

        if (thermo.get_switch() == Thermo_type::Disabled)
        {
            // Start with retrieving the stability information
            Diff_smag2_kernels::evisc_neutral_g<TF><<<gridGPU, blockGPU>>>(
                fields.sd.at("evisc")->fld_g,
                z0m_g, gd.z_g, mlen_g,
                gd.istart, gd.jstart, gd.kstart,
                gd.iend,   gd.jend,   gd.kend,
                gd.icells, gd.ijcells);
            cuda_check_error();
        }
        else
        {
            // Assume buoyancy calculation is needed
            auto tmp1 = fields.get_tmp_g();
            thermo.get_thermo_field_g(*tmp1, "N2", false);

            // Get MO gradient buoyancy:
            auto& dbdz_g  = boundary.get_dbdz_g();

            // Calculate eddy viscosity
            TF tPri = 1./tPr;

            launch_grid_kernel<Diff_smag2_kernels::evisc_g<TF, true>>(
                grid_layout,
                fields.sd.at("evisc")->fld_g.view(),
                tmp1->fld_g, dbdz_g,
                mlen_g, z0m_g, gd.z_g,
                tPri);

            fields.release_tmp_g(tmp1);
        }

        boundary_cyclic.exec_g(fields.sd.at("evisc")->fld_g);
    }
    // Do not use surface model.
    else
    {
        // Calculate total strain rate
        launch_grid_kernel<Diff_les_kernels::calc_strain2_g<TF, false>>(
            grid_layout,
            fields.sd.at("evisc")->fld_g.view(),
            fields.mp.at("u")->fld_g,
            fields.mp.at("v")->fld_g,
            fields.mp.at("w")->fld_g,
            nullptr, nullptr,
            gd.dzi_g, gd.dzhi_g,
            gd.dxi, gd.dyi);

        // start with retrieving the stability information
        if (thermo.get_switch() == Thermo_type::Disabled)
        {
            Diff_smag2_kernels::evisc_neutral_vandriest_g<TF><<<gridGPU, blockGPU>>>(
                fields.sd.at("evisc")->fld_g,
                fields.mp.at("u")->fld_g,
                fields.mp.at("v")->fld_g,
                mlen_g, gd.z_g, gd.dzhi_g,
                gd.zsize, fields.visc,
                gd.istart, gd.jstart, gd.kstart,
                gd.iend, gd.jend, gd.kend,
                gd.icells, gd.ijcells);
            cuda_check_error();
        }
        // assume buoyancy calculation is needed
        else
        {
            // store the buoyancyflux in datafluxbot of tmp1
            auto tmp1 = fields.get_tmp_g();
            thermo.get_buoyancy_fluxbot_g(*tmp1);
            // As we only use the fluxbot field of tmp1 we store the N2 in the interior.
            thermo.get_thermo_field_g(*tmp1, "N2", false);

            // Calculate eddy viscosity
            TF tPri = 1./tPr;

            launch_grid_kernel<Diff_smag2_kernels::evisc_g<TF, true>>(
                grid_layout,
                fields.sd.at("evisc")->fld_g.view(),
                tmp1->fld_g, nullptr,
                mlen_g, nullptr, gd.z_g,
                tPri);

            fields.release_tmp_g(tmp1);
        }

        boundary_cyclic.exec_g(fields.sd.at("evisc")->fld_g);

        dk::calc_ghostcells_evisc<TF><<<grid2dGPU, block2dGPU>>>(
                fields.sd.at("evisc")->fld_g,
                gd.icells, gd.jcells,
                gd.kstart, gd.kend,
                gd.icells, gd.ijcells);
    }

    cuda_check_error();
}
#endif

#ifdef USECUDA
template<typename TF>
void Diff_smag2<TF>::exec(Stats<TF>& stats)
{
    auto& gd = grid.get_grid_data();

    // Grid layout struct for cuda launcher.
    Grid_layout grid_layout = {
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.kstart, gd.kend,
            gd.istride,
            gd.jstride,
            gd.kstride};

    const TF dxidxi = TF(1)/(gd.dx * gd.dx);
    const TF dyidyi = TF(1)/(gd.dy * gd.dy);
    const TF tPri = TF(1)/tPr;

    // Do not use surface model.
    if (boundary.get_switch() == "default")
    {
        launch_grid_kernel<Diff_les_kernels::diff_uvw_g<TF, false>>(
                grid_layout,
                fields.mt.at("u")->fld_g.view(),
                fields.mt.at("v")->fld_g.view(),
                fields.mt.at("w")->fld_g.view(),
                fields.sd.at("evisc")->fld_g,
                fields.mp.at("u")->fld_g,
                fields.mp.at("v")->fld_g,
                fields.mp.at("w")->fld_g,
                fields.mp.at("u")->flux_bot_g,
                fields.mp.at("u")->flux_top_g,
                fields.mp.at("v")->flux_bot_g,
                fields.mp.at("v")->flux_top_g,
                gd.dzi_g, gd.dzhi_g,
                gd.dxi, gd.dyi,
                fields.rhoref_g, fields.rhorefh_g,
                fields.rhorefi_g, fields.rhorefhi_g,
                fields.visc);

        cuda_check_error();

        for (auto it : fields.st)
        {
            launch_grid_kernel<Diff_les_kernels::diff_c_g<TF, false>>(
                    grid_layout,
                    it.second->fld_g.view(),
                    fields.sp.at(it.first)->fld_g,
                    fields.sd.at("evisc")->fld_g,
                    fields.sp.at(it.first)->flux_bot_g,
                    fields.sp.at(it.first)->flux_top_g,
                    gd.dzi_g, gd.dzhi_g,
                    dxidxi, dyidyi,
                    fields.rhorefi_g, fields.rhorefh_g,
                    tPri, fields.sp.at(it.first)->visc);

            cuda_check_error();
        }
    }
    // Use surface model.
    else
    {
        launch_grid_kernel<Diff_les_kernels::diff_uvw_g<TF, true>>(
                grid_layout,
                fields.mt.at("u")->fld_g.view(),
                fields.mt.at("v")->fld_g.view(),
                fields.mt.at("w")->fld_g.view(),
                fields.sd.at("evisc")->fld_g,
                fields.mp.at("u")->fld_g,
                fields.mp.at("v")->fld_g,
                fields.mp.at("w")->fld_g,
                fields.mp.at("u")->flux_bot_g,
                fields.mp.at("u")->flux_top_g,
                fields.mp.at("v")->flux_bot_g,
                fields.mp.at("v")->flux_top_g,
                gd.dzi_g, gd.dzhi_g,
                gd.dxi, gd.dyi,
                fields.rhoref_g, fields.rhorefh_g,
                fields.rhorefi_g, fields.rhorefhi_g,
                fields.visc);

            cuda_check_error();

        for (auto it : fields.st)
        {
            launch_grid_kernel<Diff_les_kernels::diff_c_g<TF, true>>(
                    grid_layout,
                    it.second->fld_g.view(),
                    fields.sp.at(it.first)->fld_g,
                    fields.sd.at("evisc")->fld_g,
                    fields.sp.at(it.first)->flux_bot_g,
                    fields.sp.at(it.first)->flux_top_g,
                    gd.dzi_g, gd.dzhi_g,
                    dxidxi, dyidyi,
                    fields.rhorefi_g, fields.rhorefh_g,
                    tPri, fields.sp.at(it.first)->visc);

            cuda_check_error();
        }
    }

    hipDeviceSynchronize();
    stats.calc_tend(*fields.mt.at("u"), tend_name);
    stats.calc_tend(*fields.mt.at("v"), tend_name);
    stats.calc_tend(*fields.mt.at("w"), tend_name);
    for (auto it : fields.st)
        stats.calc_tend(*it.second, tend_name);
}
#endif

#ifdef USECUDA
template<typename TF>
unsigned long Diff_smag2<TF>::get_time_limit(unsigned long idt, double dt)
{
    namespace dk = Diff_kernels_g;
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kmax);
    dim3 blockGPU(blocki, blockj, 1);

    const TF dxidxi = TF(1)/(gd.dx * gd.dx);
    const TF dyidyi = TF(1)/(gd.dy * gd.dy);
    const TF tPrfac_i = TF(1)/std::min(TF(1.), tPr);

    auto tmp1 = fields.get_tmp_g();

    // Calculate dnmul in tmp1 field
    dk::calc_dnmul_g<TF><<<gridGPU, blockGPU>>>(
            tmp1->fld_g,
            fields.sd.at("evisc")->fld_g,
            gd.dzi_g,
            tPrfac_i,
            dxidxi, dyidyi,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.kstart, gd.kend,
            gd.icells, gd.ijcells);

    cuda_check_error();

    // Get maximum from tmp1 field
    double dnmul = field3d_operators.calc_max_g(tmp1->fld_g);
    dnmul = std::max(Constants::dsmall, dnmul);

    const unsigned long idtlim = idt * dnmax/(dnmul*dt);

    fields.release_tmp_g(tmp1);

    return idtlim;
}
#endif

#ifdef USECUDA
template<typename TF>
double Diff_smag2<TF>::get_dn(double dt)
{
    namespace dk = Diff_kernels_g;
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kmax);
    dim3 blockGPU(blocki, blockj, 1);

    const TF dxidxi = TF(1)/(gd.dx * gd.dx);
    const TF dyidyi = TF(1)/(gd.dy * gd.dy);
    const TF tPrfac_i = TF(1)/std::min(TF(1.), tPr);

    // Calculate dnmul in tmp1 field
    auto dnmul_tmp = fields.get_tmp_g();

    dk::calc_dnmul_g<TF><<<gridGPU, blockGPU>>>(
            dnmul_tmp->fld_g,
            fields.sd.at("evisc")->fld_g,
            gd.dzi_g,
            tPrfac_i,
            dxidxi, dyidyi,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.kstart, gd.kend,
            gd.icells, gd.ijcells);

    cuda_check_error();

    // Get maximum from tmp1 field
    // CvH This is odd, because there might be need for calc_max in CPU version.
    double dnmul = field3d_operators.calc_max_g(dnmul_tmp->fld_g);

    fields.release_tmp_g(dnmul_tmp);

    return dnmul*dt;
}
#endif


#ifdef FLOAT_SINGLE
template class Diff_smag2<float>;
#else
template class Diff_smag2<double>;
#endif

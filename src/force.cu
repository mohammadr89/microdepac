#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2023 Chiel van Heerwaarden
 * Copyright (c) 2011-2023 Thijs Heus
 * Copyright (c) 2014-2023 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <algorithm>
#include "master.h"
#include "grid.h"
#include "fields.h"
#include "field3d_operators.h"
#include "timeloop.h"
#include "timedep.h"
#include "stats.h"
#include "finite_difference.h"
#include "constants.h"
#include "tools.h"
#include "boundary.h"
#include "thermo.h"
#include "force.h"

// Kernel/CUDA launcher:
#include "force_kernels.cuh"
#include "cuda_launcher.h"
#include "cuda_tiling.h"

using namespace Finite_difference::O2;

namespace
{
    template<typename TF> __global__
    void add_pressure_force_g(TF* const __restrict__ ut,
                       const TF fbody,
                       const int jj, const int kk,
                       const int istart, const int jstart, const int kstart,
                       const int iend,   const int jend,   const int kend)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            ut[ijk] += fbody;
        }
    }

    template<typename TF> __global__
    void coriolis_4th_g(TF* const __restrict__ ut, TF* const __restrict__ vt,
                        TF* const __restrict__ u,  TF* const __restrict__ v,
                        TF* const __restrict__ ug, TF* const __restrict__ vg,
                        const TF fc, const TF ugrid, const TF vgrid,
                        const int jj, const int kk,
                        const int istart, const int jstart, const int kstart,
                        const int iend,   const int jend,   const int kend)
    {
        using namespace Finite_difference::O4;

        const int i   = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j   = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k   = blockIdx.z + kstart;
        const int ii  = 1;
        const int ii2 = 2;
        const int jj2 = 2*jj;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            ut[ijk] += fc * ( ( ci0<TF>*(ci0<TF>*v[ijk-ii2-jj ] + ci1<TF>*v[ijk-ii-jj ] + ci2<TF>*v[ijk-jj    ] + ci3<TF>*v[ijk+ii-jj  ])
                              + ci1<TF>*(ci0<TF>*v[ijk-ii2    ] + ci1<TF>*v[ijk-ii    ] + ci2<TF>*v[ijk       ] + ci3<TF>*v[ijk+ii     ])
                              + ci2<TF>*(ci0<TF>*v[ijk-ii2+jj ] + ci1<TF>*v[ijk-ii+jj ] + ci2<TF>*v[ijk+jj    ] + ci3<TF>*v[ijk+ii+jj  ])
                              + ci3<TF>*(ci0<TF>*v[ijk-ii2+jj2] + ci1<TF>*v[ijk-ii+jj2] + ci2<TF>*v[ijk+jj2   ] + ci3<TF>*v[ijk+ii+jj2 ]) )
                       + vgrid - vg[k] );

            vt[ijk] -= fc * ( ( ci0<TF>*(ci0<TF>*u[ijk-ii-jj2 ] + ci1<TF>*u[ijk-jj2   ] + ci2<TF>*u[ijk+ii-jj2] + ci3<TF>*u[ijk+ii2-jj2])
                              + ci1<TF>*(ci0<TF>*u[ijk-ii-jj  ] + ci1<TF>*u[ijk-jj    ] + ci2<TF>*u[ijk+ii-jj ] + ci3<TF>*u[ijk+ii2-jj ])
                              + ci2<TF>*(ci0<TF>*u[ijk-ii     ] + ci1<TF>*u[ijk       ] + ci2<TF>*u[ijk+ii    ] + ci3<TF>*u[ijk+ii2    ])
                              + ci3<TF>*(ci0<TF>*u[ijk-ii+jj  ] + ci1<TF>*u[ijk+jj    ] + ci2<TF>*u[ijk+ii+jj ] + ci3<TF>*u[ijk+ii2+jj ]) )
                       + ugrid - ug[k]);
        }
    }


    template<typename TF> __global__
    void nudging_tendency_g(
        TF* const __restrict__ st,
        const TF* const __restrict__ smn,
        const TF* const __restrict__ snudge,
        const TF* const __restrict__ nudge_fac,
        const int kstart, const int kend)
    {
        const int k = blockIdx.x*blockDim.x + threadIdx.x + kstart;

        if (k < kend)
            st[k] = -nudge_fac[k] * (smn[k]-snudge[k]);
    }


    template<typename TF>
    int calc_zi(
            const TF* const restrict fldmean,
            const int kstart, const int kend,
            const int plusminus)
    {
        TF maxgrad = 0.;
        TF grad = 0.;
        int kinv = kstart;
        for (int k=kstart+1; k<kend; ++k)
        {
            grad = plusminus * (fldmean[k] - fldmean[k-1]);
            if (grad > maxgrad)
            {
                maxgrad = grad;
                kinv = k;
            }
        }
        return kinv;
    }

    template<typename TF>
    void rescale_nudgeprof(TF* const restrict fldmean, const int kinv, const int kstart, const int kend)
    {
        for (int k=kstart+1; k<kinv; ++k)
            fldmean[k] = fldmean[kstart];

        for (int k=kinv+1; k<kend-2; ++k)
            fldmean[k] = fldmean[kend-1];
    }

    template<typename TF> __global__
    void calc_time_dependent_prof_g(
            TF* const __restrict__ prof, const TF* const __restrict__ data,
            const double fac0, const double fac1,
            const int index0,  const int index1,
            const int kmax,    const int kgc)
    {
        const int k = blockIdx.x*blockDim.x + threadIdx.x;
        const int kk = kmax;

        if (k < kmax)
            prof[k+kgc] = fac0*data[index0*kk+k] + fac1*data[index1*kk+k];
    }
} // end namespace

template<typename TF>
void Force<TF>::prepare_device()
{
    auto& gd = grid.get_grid_data();

    const int nmemsize  = gd.kcells*sizeof(TF);

    if (swlspres == Large_scale_pressure_type::Geo_wind)
    {
        ug_g.allocate(gd.kcells);
        vg_g.allocate(gd.kcells);

        cuda_safe_call(hipMemcpy(ug_g, ug.data(), nmemsize, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(vg_g, vg.data(), nmemsize, hipMemcpyHostToDevice));
    }

    if (swls == Large_scale_tendency_type::Enabled)
    {
        for (auto& it : lslist)
        {
            lsprofs_g.emplace(it, cuda_vector<TF>(gd.kcells));
            cuda_safe_call(hipMemcpy(lsprofs_g.at(it), lsprofs.at(it).data(), nmemsize, hipMemcpyHostToDevice));
        }
    }

    if (swnudge == Nudging_type::Enabled)
    {
        for (auto& it : nudgelist)
        {
            nudgeprofs_g.emplace(it, cuda_vector<TF>(gd.kcells));
            cuda_safe_call(hipMemcpy(nudgeprofs_g.at(it), nudgeprofs.at(it).data(), nmemsize, hipMemcpyHostToDevice));
        }
        nudge_factor_g.allocate(gd.kcells);
        nudge_tend_g.allocate(gd.kcells);
        cuda_safe_call(hipMemcpy(nudge_factor_g, nudge_factor.data(), nmemsize, hipMemcpyHostToDevice));
    }

    if (swwls == Large_scale_subsidence_type::Mean_field ||
        swwls == Large_scale_subsidence_type::Local_field)
    {
        wls_g.allocate(gd.kcells);
        cuda_safe_call(hipMemcpy(wls_g, wls.data(), nmemsize, hipMemcpyHostToDevice));
    }
}

template<typename TF>
void Force<TF>::clear_device()
{
    if (swlspres == Large_scale_pressure_type::Geo_wind)
    {
        for (auto& it : tdep_geo)
            it.second->clear_device();
    }

    if (swls == Large_scale_tendency_type::Enabled)
    {
        for (auto& it : tdep_ls)
            it.second->clear_device();
    }

    if (swnudge == Nudging_type::Enabled)
    {
        for (auto& it : tdep_nudge)
            it.second->clear_device();
    }

    if (swwls == Large_scale_subsidence_type::Mean_field ||
        swwls == Large_scale_subsidence_type::Local_field)
    {
        tdep_wls->clear_device();
    }
}

#ifdef USECUDA
template<typename TF>
void Force<TF>::exec(double dt, Thermo<TF>& thermo, Stats<TF>& stats)
{
    auto& gd = grid.get_grid_data();

    // Grid layout for KL/CL launches over interior.
    Grid_layout grid_layout = {
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.kstart, gd.kend,
            gd.istride,
            gd.jstride,
            gd.kstride};

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kcells);
    dim3 blockGPU(blocki, blockj, 1);

    if (swlspres == Large_scale_pressure_type::Fixed_flux)
    {
        auto tmp = fields.get_tmp_g();

        TF uavg  = field3d_operators.calc_mean_g(fields.mp.at("u")->fld_g);
        TF utavg = field3d_operators.calc_mean_g(fields.mt.at("u")->fld_g);

        fields.release_tmp_g(tmp);

        const TF fbody = (uflux - uavg - gd.utrans) / dt - utavg;

        add_pressure_force_g<TF><<<gridGPU, blockGPU>>>(
            fields.mt.at("u")->fld_g,
            fbody,
            gd.icells, gd.ijcells,
            gd.istart, gd.jstart, gd.kstart,
            gd.iend,   gd.jend,   gd.kend);
        cuda_check_error();
        hipDeviceSynchronize();

        stats.calc_tend(*fields.mt.at("u"), tend_name_pres);
    }
    else if (swlspres == Large_scale_pressure_type::Pressure_gradient)
    {
        const TF fbody = TF(-1.)*dpdx;
        add_pressure_force_g<TF><<<gridGPU, blockGPU>>>(
            fields.mt.at("u")->fld_g, fbody,
            gd.icells, gd.ijcells,
            gd.istart, gd.jstart, gd.kstart,
            gd.iend,   gd.jend,   gd.kend);
        cuda_check_error();
        hipDeviceSynchronize();

        stats.calc_tend(*fields.mt.at("u"), tend_name_pres);
    }
    else if (swlspres == Large_scale_pressure_type::Geo_wind)
    {
        TF fc_loc = fc;
        if (fc_loc < 0)
            fc_loc = 2. * Constants::e_rot<TF> * std::sin(gd.lat * TF(M_PI) / 180.);
        
        if (grid.get_spatial_order() == Grid_order::Second)
        {
            launch_grid_kernel<Force_kernels::coriolis_2nd_g<TF>>(
                    grid_layout,
                    fields.mt.at("u")->fld_g.view(),
                    fields.mt.at("v")->fld_g.view(),
                    fields.mp.at("u")->fld_g,
                    fields.mp.at("v")->fld_g,
                    ug_g, vg_g, fc,
                    gd.utrans, gd.vtrans);
        }
        else if (grid.get_spatial_order() == Grid_order::Fourth)
        {
            coriolis_4th_g<TF><<<gridGPU, blockGPU>>>(
                fields.mt.at("u")->fld_g, fields.mt.at("v")->fld_g,
                fields.mp.at("u")->fld_g, fields.mp.at("v")->fld_g,
                ug_g, vg_g, fc, gd.utrans, gd.vtrans,
                gd.icells, gd.ijcells,
                gd.istart, gd.jstart, gd.kstart,
                gd.iend,   gd.jend,   gd.kend);
            cuda_check_error();
        }
        hipDeviceSynchronize();

        stats.calc_tend(*fields.mt.at("u"), tend_name_cor);
        stats.calc_tend(*fields.mt.at("v"), tend_name_cor);
    }

    if (swls == Large_scale_tendency_type::Enabled)
    {
        for (auto& it : lslist)
        {
            launch_grid_kernel<Force_kernels::add_profile_g<TF>>(
                    grid_layout,
                    fields.at.at(it)->fld_g.view(),
                    lsprofs_g.at(it));

            hipDeviceSynchronize();
            stats.calc_tend(*fields.at.at(it), tend_name_ls);
        }
    }

    if (swnudge == Nudging_type::Enabled)
    {
        for (auto& it : nudgelist)
        {
            auto it1 = std::find(scalednudgelist.begin(), scalednudgelist.end(), it);
            if (it1 != scalednudgelist.end())
            {
                hipMemcpy(fields.ap.at(it)->fld_mean.data(), fields.ap.at(it)->fld_mean_g, gd.kcells*sizeof(TF), hipMemcpyDeviceToHost);
                const int kinv = thermo.get_bl_depth();
                rescale_nudgeprof(nudgeprofs.at(it).data(), kinv, gd.kstart, gd.kend);
                hipMemcpy(nudgeprofs_g.at(it), nudgeprofs.at(it).data(), gd.kcells*sizeof(TF), hipMemcpyHostToDevice);
            }

            // Calculate nudging tendency profile.
            const int blocki = 32;
            const int gridi  = gd.kmax/blocki + (gd.kmax%blocki > 0);

            dim3 gridGPU (gridi);
            dim3 blockGPU(blocki);

            // Calculate nudging tendency as single profile.
            nudging_tendency_g<TF><<<gridGPU, blockGPU>>>(
                    nudge_tend_g.view(),
                    fields.ap.at(it)->fld_mean_g,
                    nudgeprofs_g.at(it),
                    nudge_factor_g,
                    gd.kstart, gd.kend);

            // Add tendency profile to 3D tendency field.
            launch_grid_kernel<Force_kernels::add_profile_g<TF>>(
                    grid_layout,
                    fields.at.at(it)->fld_g.view(),
                    nudge_tend_g);

            hipDeviceSynchronize();
            stats.calc_tend(*fields.at.at(it), tend_name_nudge);
        }
    }

    if (swwls == Large_scale_subsidence_type::Mean_field)
    {
        if (swwls_mom)
        {
            launch_grid_kernel<Force_kernels::advec_wls_2nd_mean_g<TF>>(
                    grid_layout,
                    fields.mt.at("u")->fld_g.view(),
                    fields.mp.at("u")->fld_mean_g,
                    wls_g, gd.dzhi_g);

            hipDeviceSynchronize();
            stats.calc_tend(*fields.mt.at("u"), tend_name_subs);

            launch_grid_kernel<Force_kernels::advec_wls_2nd_mean_g<TF>>(
                    grid_layout,
                    fields.mt.at("v")->fld_g.view(),
                    fields.mp.at("v")->fld_mean_g,
                    wls_g, gd.dzhi_g);

            hipDeviceSynchronize();
            stats.calc_tend(*fields.mt.at("v"), tend_name_subs);
        }

        for (auto& it : fields.st)
        {
            launch_grid_kernel<Force_kernels::advec_wls_2nd_mean_g<TF>>(
                    grid_layout,
                    fields.st.at(it.first)->fld_g.view(),
                    fields.sp.at(it.first)->fld_mean_g,
                    wls_g, gd.dzhi_g);

            hipDeviceSynchronize();
            stats.calc_tend(*it.second, tend_name_subs);
        }
    }
    else if (swwls == Large_scale_subsidence_type::Local_field)
    {
        if (swwls_mom)
        {
            launch_grid_kernel<Force_kernels::advec_wls_2nd_local_g<TF>>(
                    grid_layout,
                    fields.mt.at("u")->fld_g.view(),
                    fields.mp.at("u")->fld_g,
                    wls_g, gd.dzhi_g);

            hipDeviceSynchronize();
            stats.calc_tend(*fields.mt.at("u"), tend_name_subs);

            launch_grid_kernel<Force_kernels::advec_wls_2nd_local_g<TF>>(
                    grid_layout,
                    fields.mt.at("v")->fld_g.view(),
                    fields.mp.at("v")->fld_g,
                    wls_g, gd.dzhi_g);

            hipDeviceSynchronize();
            stats.calc_tend(*fields.mt.at("v"), tend_name_subs);

            // Modified `grid_layout`, which starts at kstart+1 (for w subsidence).
            Grid_layout grid_layout_kp1 = {
                    gd.istart,   gd.iend,
                    gd.jstart,   gd.jend,
                    gd.kstart+1, gd.kend,
                    gd.istride,
                    gd.jstride,
                    gd.kstride};

            launch_grid_kernel<Force_kernels::advec_wls_2nd_local_w_g<TF>>(
                    grid_layout_kp1,
                    fields.mt.at("w")->fld_g.view(),
                    fields.mp.at("w")->fld_g,
                    wls_g, gd.dzi_g);

            hipDeviceSynchronize();
            stats.calc_tend(*fields.mt.at("w"), tend_name_subs);
        }

        for (auto& it : fields.st)
        {
            launch_grid_kernel<Force_kernels::advec_wls_2nd_local_g<TF>>(
                    grid_layout,
                    fields.st.at(it.first)->fld_g.view(),
                    fields.sp.at(it.first)->fld_g,
                    wls_g, gd.dzhi_g);

            hipDeviceSynchronize();
            stats.calc_tend(*it.second, tend_name_subs);
        }
    }
}
#endif

#ifdef USECUDA
template <typename TF>
void Force<TF>::update_time_dependent(Timeloop<TF>& timeloop)
{
    if (swls == Large_scale_tendency_type::Enabled)
    {
        for (auto& it : tdep_ls)
            it.second->update_time_dependent_prof_g(lsprofs_g.at(it.first), timeloop);
    }

    if (swnudge == Nudging_type::Enabled)
    {
        for (auto& it : tdep_nudge)
            it.second->update_time_dependent_prof_g(nudgeprofs_g.at(it.first), timeloop);
    }

    if (swlspres == Large_scale_pressure_type::Geo_wind)
    {
        tdep_geo.at("u_geo")->update_time_dependent_prof_g(ug_g, timeloop);
        tdep_geo.at("v_geo")->update_time_dependent_prof_g(vg_g, timeloop);
    }

    if (swwls == Large_scale_subsidence_type::Mean_field ||
        swwls == Large_scale_subsidence_type::Local_field)
    {
        tdep_wls->update_time_dependent_prof_g(wls_g, timeloop);
    }
}
#endif


#ifdef FLOAT_SINGLE
template class Force<float>;
#else
template class Force<double>;
#endif

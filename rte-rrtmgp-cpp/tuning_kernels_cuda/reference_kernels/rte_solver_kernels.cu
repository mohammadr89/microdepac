#include "hip/hip_runtime.h"
#include "Types.h"

template<typename TF> __device__ constexpr TF k_min();
template<> __device__ constexpr double k_min() { return 1.e-12; }
template<> __device__ constexpr float k_min() { return 1.e-4f; }

template<typename TF> __device__
void lw_source_noscat_kernel(
        const int icol, const int ilay, const int igpt, const int ncol, const int nlay, const int ngpt, const TF eps,
        const TF* __restrict__ lay_source, const TF* __restrict__ lev_source_up, const TF* __restrict__ lev_source_dn,
        const TF* __restrict__ tau, const TF* __restrict__ trans, TF* __restrict__ source_dn, TF* __restrict__ source_up)
{
    const TF tau_thres = sqrt(eps);

    const int idx = icol + ilay*ncol + igpt*ncol*nlay;
    const TF fact = (tau[idx]>tau_thres) ? (TF(1.) - trans[idx]) / tau[idx] - trans[idx] : tau[idx] * (TF(.5) - TF(1.)/TF(3.)*tau[idx]);
    source_dn[idx] = (TF(1.) - trans[idx]) * lev_source_dn[idx] + TF(2.) * fact * (lay_source[idx]-lev_source_dn[idx]);
    source_up[idx] = (TF(1.) - trans[idx]) * lev_source_up[idx] + TF(2.) * fact * (lay_source[idx]-lev_source_up[idx]);
}

template<typename TF>__device__
void lw_transport_noscat_kernel(
        const int icol, const int igpt, const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1,
        const TF* __restrict__ tau, const TF* __restrict__ trans, const TF* __restrict__ sfc_albedo,
        const TF* __restrict__ source_dn, const TF* __restrict__ source_up, const TF* __restrict__ source_sfc,
        TF* __restrict__ radn_up, TF* __restrict__ radn_dn, const TF* __restrict__ source_sfc_jac, TF* __restrict__ radn_up_jac)
{
    if (top_at_1)
    {
        for (int ilev=1; ilev<(nlay+1); ++ilev)
        {
            const int idx1 = icol + ilev*ncol + igpt*ncol*(nlay+1);
            const int idx2 = icol + (ilev-1)*ncol + igpt*ncol*(nlay+1);
            const int idx3 = icol + (ilev-1)*ncol + igpt*ncol*nlay;
            radn_dn[idx1] = trans[idx3] * radn_dn[idx2] + source_dn[idx3];
        }

        const int idx_bot = icol + nlay*ncol + igpt*ncol*(nlay+1);
        const int idx2d = icol + igpt*ncol;
        radn_up[idx_bot] = radn_dn[idx_bot] * sfc_albedo[idx2d] + source_sfc[idx2d];
        radn_up_jac[idx_bot] = source_sfc_jac[idx2d];

        for (int ilev=nlay-1; ilev>=0; --ilev)
        {
            const int idx1 = icol + ilev*ncol + igpt*ncol*(nlay+1);
            const int idx2 = icol + (ilev+1)*ncol + igpt*ncol*(nlay+1);
            const int idx3 = icol + ilev*ncol + igpt*ncol*nlay;
            radn_up[idx1] = trans[idx3] * radn_up[idx2] + source_up[idx3];
            radn_up_jac[idx1] = trans[idx3] * radn_up_jac[idx2];
        }
    }
    else
    {
        for (int ilev=(nlay-1); ilev>=0; --ilev)
        {
            const int idx1 = icol + ilev*ncol + igpt*ncol*(nlay+1);
            const int idx2 = icol + (ilev+1)*ncol + igpt*ncol*(nlay+1);
            const int idx3 = icol + ilev*ncol + igpt*ncol*nlay;
            radn_dn[idx1] = trans[idx3] * radn_dn[idx2] + source_dn[idx3];
        }

        const int idx_bot = icol + igpt*ncol*(nlay+1);
        const int idx2d = icol + igpt*ncol;
        radn_up[idx_bot] = radn_dn[idx_bot] * sfc_albedo[idx2d] + source_sfc[idx2d];
        radn_up_jac[idx_bot] = source_sfc_jac[idx2d];

        for (int ilev=1; ilev<(nlay+1); ++ilev)
        {
            const int idx1 = icol + ilev*ncol + igpt*ncol*(nlay+1);
            const int idx2 = icol + (ilev-1)*ncol + igpt*ncol*(nlay+1);
            const int idx3 = icol + (ilev-1)*ncol + igpt*ncol*nlay;
            radn_up[idx1] = trans[idx3] * radn_up[idx2] + source_up[idx3];
            radn_up_jac[idx1] = trans[idx3] * radn_up_jac[idx2];;
        }
    }
}


template<typename TF> __global__
void lw_solver_noscat_step1_kernel(
        const int ncol, const int nlay, const int ngpt, const TF eps, const BOOL_TYPE top_at_1,
        const TF* __restrict__ D, const TF* __restrict__ weight, const TF* __restrict__ tau, const TF* __restrict__ lay_source,
        const TF* __restrict__ lev_source_inc, const TF* __restrict__ lev_source_dec, const TF* __restrict__ sfc_emis,
        const TF* __restrict__ sfc_src, TF* __restrict__ radn_up, TF* __restrict__ radn_dn,
        const TF* __restrict__ sfc_src_jac, TF* __restrict__ radn_up_jac, TF* __restrict__ tau_loc,
        TF* __restrict__ trans, TF* __restrict__ source_dn, TF* __restrict__ source_up,
        TF* __restrict__ source_sfc, TF* __restrict__ sfc_albedo, TF* __restrict__ source_sfc_jac)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
    const int igpt = blockIdx.z*blockDim.z + threadIdx.z;

    if ( (icol < ncol) && (ilay < nlay) && (igpt < ngpt) )
    {
        const TF pi = acos(TF(-1.));
        const TF* lev_source_up;
        const TF* lev_source_dn;
        int top_level;

        if (top_at_1)
        {
            top_level = 0;
            lev_source_up = lev_source_dec;
            lev_source_dn = lev_source_inc;
        }
        else
        {
            top_level = nlay;
            lev_source_up = lev_source_inc;
            lev_source_dn = lev_source_dec;
        }

        if ( blockIdx.y == 0 && threadIdx.y == 0 )
        {
            const int idx_top = icol + top_level*ncol + igpt*ncol*(nlay+1);
            radn_dn[idx_top] = radn_dn[idx_top] / (TF(2.) * pi * weight[0]);
        }

        const int idx2d = icol + igpt*ncol;

        const int idx3d = icol + ilay*ncol + igpt*ncol*nlay;
        tau_loc[idx3d] = tau[idx3d] * D[0];
        trans[idx3d] = exp(-tau_loc[idx3d]);

        lw_source_noscat_kernel(
                icol, ilay, igpt, ncol, nlay, ngpt, eps, lay_source, lev_source_up, lev_source_dn,
                tau_loc, trans, source_dn, source_up);

        sfc_albedo[idx2d] = TF(1.) - sfc_emis[idx2d];
        source_sfc[idx2d] = sfc_emis[idx2d] * sfc_src[idx2d];
        source_sfc_jac[idx2d] = sfc_emis[idx2d] * sfc_src_jac[idx2d];
    }
}


template<typename TF> __global__
void lw_solver_noscat_step2_kernel(
        const int ncol, const int nlay, const int ngpt, const TF eps, const BOOL_TYPE top_at_1,
        const TF* __restrict__ D, const TF* __restrict__ weight, const TF* __restrict__ tau, const TF* __restrict__ lay_source,
        const TF* __restrict__ lev_source_inc, const TF* __restrict__ lev_source_dec, const TF* __restrict__ sfc_emis,
        const TF* __restrict__ sfc_src, TF* __restrict__ radn_up, TF* __restrict__ radn_dn,
        const TF* __restrict__ sfc_src_jac, TF* __restrict__ radn_up_jac, TF* __restrict__ tau_loc,
        TF* __restrict__ trans, TF* __restrict__ source_dn, TF* __restrict__ source_up,
        TF* __restrict__ source_sfc, TF* __restrict__ sfc_albedo, TF* __restrict__ source_sfc_jac)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int igpt = blockIdx.y*blockDim.y + threadIdx.y;

    if ( (icol < ncol) && (igpt < ngpt) )
    {
        lw_transport_noscat_kernel(
                icol, igpt, ncol, nlay, ngpt, top_at_1, tau, trans, sfc_albedo, source_dn,
                source_up, source_sfc, radn_up, radn_dn, source_sfc_jac, radn_up_jac);
    }
}


template<typename TF> __global__
void lw_solver_noscat_step3_kernel(
        const int ncol, const int nlay, const int ngpt, const TF eps, const BOOL_TYPE top_at_1,
        const TF* __restrict__ D, const TF* __restrict__ weight, const TF* __restrict__ tau, const TF* __restrict__ lay_source,
        const TF* __restrict__ lev_source_inc, const TF* __restrict__ lev_source_dec, const TF* __restrict__ sfc_emis,
        const TF* __restrict__ sfc_src, TF* __restrict__ radn_up, TF* __restrict__ radn_dn,
        const TF* __restrict__ sfc_src_jac, TF* __restrict__ radn_up_jac, TF* __restrict__ tau_loc,
        TF* __restrict__ trans, TF* __restrict__ source_dn, TF* __restrict__ source_up,
        TF* __restrict__ source_sfc, TF* __restrict__ sfc_albedo, TF* __restrict__ source_sfc_jac)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int ilev = blockIdx.y*blockDim.y + threadIdx.y;
    const int igpt = blockIdx.z*blockDim.z + threadIdx.z;

    if ( (icol < ncol) && (ilev < (nlay+1)) && (igpt < ngpt) )
    {
        const TF pi = acos(TF(-1.));

        const int idx = icol + ilev*ncol + igpt*ncol*(nlay+1);
        radn_up[idx] *= TF(2.) * pi * weight[0];
        radn_dn[idx] *= TF(2.) * pi * weight[0];
        radn_up_jac[idx] *= TF(2.) * pi * weight[0];
    }
}


template<typename TF> __global__
void sw_adding_kernel(
        const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1,
        const TF* __restrict__ sfc_alb_dif, const TF* __restrict__ r_dif, const TF* __restrict__ t_dif,
        const TF* __restrict__ source_dn, const TF* __restrict__ source_up, const TF* __restrict__ source_sfc,
        TF* __restrict__ flux_up, TF* __restrict__ flux_dn, const TF* __restrict__ flux_dir,
        TF* __restrict__ albedo, TF* __restrict__ src, TF* __restrict__ denom)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int igpt = blockIdx.y*blockDim.y + threadIdx.y;

    if ( (icol < ncol) && (igpt < ngpt) )
    {
        if (top_at_1)
        {
            const int sfc_idx_3d = icol + nlay*ncol + igpt*(nlay+1)*ncol;
            const int sfc_idx_2d = icol + igpt*ncol;
            albedo[sfc_idx_3d] = sfc_alb_dif[sfc_idx_2d];
            src[sfc_idx_3d] = source_sfc[sfc_idx_2d];

            for (int ilay=nlay-1; ilay >= 0; --ilay)
            {
                const int lay_idx  = icol + ilay*ncol + igpt*ncol*nlay;
                const int lev_idx1 = icol + ilay*ncol + igpt*ncol*(nlay+1);
                const int lev_idx2 = icol + (ilay+1)*ncol + igpt*ncol*(nlay+1);
                denom[lay_idx] = TF(1.)/(TF(1.) - r_dif[lay_idx] * albedo[lev_idx2]);
                albedo[lev_idx1] = r_dif[lay_idx] + t_dif[lay_idx] * t_dif[lay_idx]
                                                  * albedo[lev_idx2] * denom[lay_idx];
                src[lev_idx1] = source_up[lay_idx] + t_dif[lay_idx] * denom[lay_idx] *
                                (src[lev_idx2] + albedo[lev_idx2] * source_dn[lay_idx]);
            }
            const int top_idx = icol + igpt*(nlay+1)*ncol;
            flux_up[top_idx] = flux_dn[top_idx]*albedo[top_idx] + src[top_idx];

            for (int ilay=1; ilay < (nlay+1); ++ilay)
            {
                const int lev_idx1 = icol + ilay*ncol + igpt*(nlay+1)*ncol;
                const int lev_idx2 = icol + (ilay-1)*ncol + igpt*(nlay+1)*ncol;
                const int lay_idx = icol + (ilay-1)*ncol + igpt*(nlay)*ncol;
                flux_dn[lev_idx1] = (t_dif[lay_idx]*flux_dn[lev_idx2] +
                                     r_dif[lay_idx]*src[lev_idx1] +
                                     source_dn[lay_idx]) * denom[lay_idx];
                flux_up[lev_idx1] = flux_dn[lev_idx1] * albedo[lev_idx1] + src[lev_idx1];
            }

            for (int ilay=0; ilay < (nlay+1); ++ilay)
            {
                const int idx = icol + ilay*ncol + igpt*(nlay+1)*ncol;
                flux_dn[idx] += flux_dir[idx];
            }
        }
        else
        {
            const int sfc_idx_3d = icol + igpt*(nlay+1)*ncol;
            const int sfc_idx_2d = icol + igpt*ncol;
            albedo[sfc_idx_3d] = sfc_alb_dif[sfc_idx_2d];
            src[sfc_idx_3d] = source_sfc[sfc_idx_2d];

            for (int ilay=0; ilay<nlay; ++ilay)
            {
                const int lay_idx  = icol + ilay*ncol + igpt*ncol*nlay;
                const int lev_idx1 = icol + ilay*ncol + igpt*ncol*(nlay+1);
                const int lev_idx2 = icol + (ilay+1)*ncol + igpt*ncol*(nlay+1);
                denom[lay_idx] = TF(1.)/(TF(1.) - r_dif[lay_idx] * albedo[lev_idx1]);
                albedo[lev_idx2] = r_dif[lay_idx] + (t_dif[lay_idx] * t_dif[lay_idx] *
                                                     albedo[lev_idx1] * denom[lay_idx]);
                src[lev_idx2] = source_up[lay_idx] + t_dif[lay_idx]*denom[lay_idx]*
                                                     (src[lev_idx1]+albedo[lev_idx1]*source_dn[lay_idx]);
            }
            const int top_idx = icol + nlay*ncol + igpt*(nlay+1)*ncol;
            flux_up[top_idx] = flux_dn[top_idx] *albedo[top_idx] + src[top_idx];

            for (int ilay=nlay-1; ilay >= 0; --ilay)
            {
                    const int lev_idx1 = icol + ilay*ncol + igpt*(nlay+1)*ncol;
                    const int lev_idx2 = icol + (ilay+1)*ncol + igpt*(nlay+1)*ncol;
                    const int lay_idx = icol + ilay*ncol + igpt*nlay*ncol;
                    flux_dn[lev_idx1] = (t_dif[lay_idx]*flux_dn[lev_idx2] +
                                         r_dif[lay_idx]*src[lev_idx1] +
                                         source_dn[lay_idx]) * denom[lay_idx];
                    flux_up[lev_idx1] = flux_dn[lev_idx1] * albedo[lev_idx1] + src[lev_idx1];
            }
            for (int ilay=nlay; ilay >= 0; --ilay)
            {
                const int idx = icol + ilay*ncol + igpt*(nlay+1)*ncol;
                flux_dn[idx] += flux_dir[idx];
            }
        }
    }
}

template<typename TF> __global__
void sw_source_kernel(
        const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1,
        TF* __restrict__ r_dir, TF* __restrict__ t_dir, TF* __restrict__ t_noscat,
        const TF* __restrict__ sfc_alb_dir, TF* __restrict__ source_up, TF* __restrict__ source_dn,
        TF* __restrict__ source_sfc, TF* __restrict__ flux_dir)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int igpt = blockIdx.y*blockDim.y + threadIdx.y;

    if ( (icol < ncol) && (igpt < ngpt) )
    {
        if (top_at_1)
        {
            for (int ilay=0; ilay<nlay; ++ilay)
            {
                const int idx_lay  = icol + ilay*ncol + igpt*nlay*ncol;
                const int idx_lev1 = icol + ilay*ncol + igpt*(nlay+1)*ncol;
                const int idx_lev2 = icol + (ilay+1)*ncol + igpt*(nlay+1)*ncol;
                source_up[idx_lay] = r_dir[idx_lay] * flux_dir[idx_lev1];
                source_dn[idx_lay] = t_dir[idx_lay] * flux_dir[idx_lev1];
                flux_dir[idx_lev2] = t_noscat[idx_lay] * flux_dir[idx_lev1];

            }
            const int sfc_idx = icol + igpt*ncol;
            const int flx_idx = icol + nlay*ncol + igpt*(nlay+1)*ncol;
            source_sfc[sfc_idx] = flux_dir[flx_idx] * sfc_alb_dir[icol];
        }
        else
        {
            for (int ilay=nlay-1; ilay>=0; --ilay)
            {
                const int idx_lay  = icol + ilay*ncol + igpt*nlay*ncol;
                const int idx_lev1 = icol + (ilay)*ncol + igpt*(nlay+1)*ncol;
                const int idx_lev2 = icol + (ilay+1)*ncol + igpt*(nlay+1)*ncol;
                source_up[idx_lay] = r_dir[idx_lay] * flux_dir[idx_lev2];
                source_dn[idx_lay] = t_dir[idx_lay] * flux_dir[idx_lev2];
                flux_dir[idx_lev1] = t_noscat[idx_lay] * flux_dir[idx_lev2];

            }
            const int sfc_idx = icol + igpt*ncol;
            const int flx_idx = icol + igpt*(nlay+1)*ncol;
            source_sfc[sfc_idx] = flux_dir[flx_idx] * sfc_alb_dir[icol];
        }
    }
}

template<typename TF> __global__
void apply_BC_kernel_lw(const int isfc, int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1, const TF* __restrict__ inc_flux, TF* __restrict__ flux_dn)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int igpt = blockIdx.y*blockDim.y + threadIdx.y;

    if ( (icol < ncol) && (igpt < ngpt) )
    {
        const int idx_in  = icol + isfc*ncol + igpt*ncol*(nlay+1);
        const int idx_out = (top_at_1) ? icol + igpt*ncol*(nlay+1) : icol + nlay*ncol + igpt*ncol*(nlay+1);
        flux_dn[idx_out] = inc_flux[idx_in];
    }
}

template<typename TF>__global__ //apply_BC_gpt
void apply_BC_kernel(const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1, const TF* __restrict__ inc_flux, TF* __restrict__ flux_dn)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int igpt = blockIdx.y*blockDim.y + threadIdx.y;
    if ( (icol < ncol) && (igpt < ngpt) )
    {
        if (top_at_1)
        {
            const int idx_out = icol + igpt*ncol*(nlay+1);
            const int idx_in  = icol + igpt*ncol;
            flux_dn[idx_out] = inc_flux[idx_in];
        }
        else
        {
            const int idx_out = icol + nlay*ncol + igpt*ncol*(nlay+1);
            const int idx_in  = icol + igpt*ncol;
            flux_dn[idx_out] = inc_flux[idx_in];
        }
    }
}

template<typename TF>__global__
void apply_BC_kernel(const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1, const TF* __restrict__ inc_flux, const TF* __restrict__ factor, TF* __restrict__ flux_dn)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int igpt = blockIdx.y*blockDim.y + threadIdx.y;
    if ( (icol < ncol) && (igpt < ngpt) )
    {
        if (top_at_1)
        {
            const int idx_out = icol + igpt*ncol*(nlay+1);
            const int idx_in  = icol + igpt*ncol;
            flux_dn[idx_out] = inc_flux[idx_in] * factor[icol];
        }
        else
        {
            const int idx_out = icol + nlay*ncol + igpt*ncol*(nlay+1);
            const int idx_in  = icol + igpt*ncol;
            flux_dn[idx_out] = inc_flux[idx_in] * factor[icol];
        }
    }
}

template<typename TF>__global__
void apply_BC_kernel(const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1, TF* __restrict__ flux_dn)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int igpt = blockIdx.y*blockDim.y + threadIdx.y;
    if ( (icol < ncol) && (igpt < ngpt) )
    {
        if (top_at_1)
        {
            const int idx_out = icol + igpt*ncol*(nlay+1);
            flux_dn[idx_out] = TF(0.);
        }
        else
        {
            const int idx_out = icol + nlay*ncol + igpt*ncol*(nlay+1);
            flux_dn[idx_out] = TF(0.);
        }
    }
}

template<typename TF>__global__
void sw_2stream_kernel(
        const int ncol, const int nlay, const int ngpt, const TF tmin,
        const TF* __restrict__ tau, const TF* __restrict__ ssa,
        const TF* __restrict__ g, const TF* __restrict__ mu0,
        TF* __restrict__ r_dif, TF* __restrict__ t_dif,
        TF* __restrict__ r_dir, TF* __restrict__ t_dir,
        TF* __restrict__ t_noscat)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int ilay = blockIdx.y*blockDim.y + threadIdx.y;
    const int igpt = blockIdx.z*blockDim.z + threadIdx.z;

    if ( (icol < ncol) && (ilay < nlay) && (igpt < ngpt) )
    {
        const int idx = icol + ilay*ncol + igpt*nlay*ncol;
        const TF mu0_inv = TF(1.)/mu0[icol];
        const TF gamma1 = (TF(8.) - ssa[idx] * (TF(5.) + TF(3.) * g[idx])) * TF(.25);
        const TF gamma2 = TF(3.) * (ssa[idx] * (TF(1.) -          g[idx])) * TF(.25);
        const TF gamma3 = (TF(2.) - TF(3.) * mu0[icol] *          g[idx])  * TF(.25);
        const TF gamma4 = TF(1.) - gamma3;

        const TF alpha1 = gamma1 * gamma4 + gamma2 * gamma3;
        const TF alpha2 = gamma1 * gamma3 + gamma2 * gamma4;

        const TF k = sqrt(max((gamma1 - gamma2) * (gamma1 + gamma2), k_min<TF>()));
        const TF exp_minusktau = exp(-tau[idx] * k);
        const TF exp_minus2ktau = exp_minusktau * exp_minusktau;

        const TF rt_term = TF(1.) / (k      * (TF(1.) + exp_minus2ktau) +
                                     gamma1 * (TF(1.) - exp_minus2ktau));
        r_dif[idx] = rt_term * gamma2 * (TF(1.) - exp_minus2ktau);
        t_dif[idx] = rt_term * TF(2.) * k * exp_minusktau;
        t_noscat[idx] = exp(-tau[idx] * mu0_inv);

        const TF k_mu     = k * mu0[icol];
        const TF k_gamma3 = k * gamma3;
        const TF k_gamma4 = k * gamma4;

        const TF fact = (abs(TF(1.) - k_mu*k_mu) > tmin) ? TF(1.) - k_mu*k_mu : tmin;
        const TF rt_term2 = ssa[idx] * rt_term / fact;

        r_dir[idx] = rt_term2  * ((TF(1.) - k_mu) * (alpha2 + k_gamma3)   -
                                  (TF(1.) + k_mu) * (alpha2 - k_gamma3) * exp_minus2ktau -
                                   TF(2.) * (k_gamma3 - alpha2 * k_mu)  * exp_minusktau * t_noscat[idx]);

        t_dir[idx] = -rt_term2 * ((TF(1.) + k_mu) * (alpha1 + k_gamma4) * t_noscat[idx]   -
                                  (TF(1.) - k_mu) * (alpha1 - k_gamma4) * exp_minus2ktau * t_noscat[idx] -
                                   TF(2.) * (k_gamma4 + alpha1 * k_mu)  * exp_minusktau);
    }
}

/*
template<typename TF>__global__
void sw_source_adding_kernel(const int ncol, const int nlay, const int ngpt, const BOOL_TYPE top_at_1,
                             const TF* __restrict__ sfc_alb_dir, const TF* __restrict__ sfc_alb_dif,
                             TF* __restrict__ r_dif, TF* __restrict__ t_dif,
                             TF* __restrict__ r_dir, TF* __restrict__ t_dir, TF* __restrict__ t_noscat,
                             TF* __restrict__ flux_up, TF* __restrict__ flux_dn, TF* __restrict__ flux_dir,
                             TF* __restrict__ source_up, TF* __restrict__ source_dn, TF* __restrict__ source_sfc,
                             TF* __restrict__ albedo, TF* __restrict__ src, TF* __restrict__ denom)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int igpt = blockIdx.y*blockDim.y + threadIdx.y;

    if ( (icol < ncol) && (igpt < ngpt) )
    {
        sw_source_kernel(icol, igpt, ncol, nlay, top_at_1, r_dir, t_dir,
                         t_noscat, sfc_alb_dir, source_up, source_dn, source_sfc, flux_dir);

        sw_adding_kernel(icol, igpt, ncol, nlay, top_at_1, sfc_alb_dif,
                         r_dif, t_dif, source_dn, source_up, source_sfc,
                         flux_up, flux_dn, flux_dir, albedo, src, denom);
    }
}
*/
template<typename TF>__global__
void lw_solver_noscat_gaussquad_kernel(const int ncol, const int nlay, const int ngpt, const TF eps,
                                       const BOOL_TYPE top_at_1, const int nmus, const TF* __restrict__ ds, const TF* __restrict__ weights,
                                       const TF* __restrict__ tau, const TF* __restrict__ lay_source,
                                       const TF* __restrict__ lev_source_inc, const TF* __restrict__ lev_source_dec, const TF* __restrict__ sfc_emis,
                                       const TF* __restrict__ sfc_src, TF* __restrict__ radn_up, TF* __restrict__ radn_dn,
                                       const TF* __restrict__ sfc_src_jac, TF* __restrict__ radn_up_jac, TF* __restrict__ tau_loc,
                                       TF* __restrict__ trans, TF* __restrict__ source_dn, TF* __restrict__ source_up,
                                       TF* __restrict__ source_sfc, TF* __restrict__ sfc_albedo, TF* __restrict__ source_sfc_jac,
                                       TF* __restrict__ flux_up, TF* __restrict__ flux_dn, TF* __restrict__ flux_up_jac)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int igpt = blockIdx.y*blockDim.y + threadIdx.y;

    if ( (icol < ncol) && (igpt < ngpt) )
    {
        lw_solver_noscat_kernel(icol, igpt, ncol, nlay, ngpt, eps, top_at_1, ds[0], weights[0], tau, lay_source,
                         lev_source_inc, lev_source_dec, sfc_emis, sfc_src, flux_up, flux_dn, sfc_src_jac,
                         flux_up_jac, tau_loc, trans, source_dn, source_up, source_sfc, sfc_albedo, source_sfc_jac);
        const int top_level = top_at_1 ? 0 : nlay;
        apply_BC_kernel_lw(icol, igpt, top_level, ncol, nlay, ngpt, top_at_1, flux_dn, radn_dn);

        if (nmus > 1)
        {
            for (int imu=1; imu<nmus; ++imu)
            {
                lw_solver_noscat_kernel(icol, igpt, ncol, nlay, ngpt, eps, top_at_1, ds[imu], weights[imu], tau, lay_source,
                                 lev_source_inc, lev_source_dec, sfc_emis, sfc_src, radn_up, radn_dn, sfc_src_jac,
                                 radn_up_jac, tau_loc, trans, source_dn, source_up, source_sfc, sfc_albedo, source_sfc_jac);

                for (int ilev=0; ilev<(nlay+1); ++ilev)
                {
                    const int idx = icol + ilev*ncol + igpt*ncol*(nlay+1);
                    flux_up[idx] += radn_up[idx];
                    flux_dn[idx] += radn_dn[idx];
                    flux_up_jac[idx] += radn_up_jac[idx];
                }
            }
        }
    }
}


template<typename TF> __global__
void add_fluxes_kernel(
        const int ncol, const int nlev, const int ngpt,
        const TF* __restrict__ radn_up, const TF* __restrict__ radn_dn, const TF* __restrict__ radn_up_jac,
        TF* __restrict__ flux_up, TF* __restrict__ flux_dn, TF* __restrict__ flux_up_jac)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int ilev = blockIdx.y*blockDim.y + threadIdx.y;
    const int igpt = blockIdx.z*blockDim.z + threadIdx.z;

    if ( (icol < ncol) && (ilev < nlev) && (igpt < ngpt) )
    {
        const int idx = icol + ilev*ncol + igpt*ncol*nlev;

        flux_up[idx] += radn_up[idx];
        flux_dn[idx] += radn_dn[idx];
        flux_up_jac[idx] += radn_up_jac[idx];
    }
}

#include "hip/hip_runtime.h"
__global__
void get_from_gpoint_kernel(const int ncol, const int igpt, Float* __restrict__ var_full, const Float* __restrict__ var_sub)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;

    if ( (icol < ncol) )
    {
        const int idx_full = icol + igpt*ncol;
        const int idx_sub = icol;
        var_full[idx_full] = var_sub[idx_sub];
    }
}

__global__
void add_from_gpoint_kernel(const int ncol, const int nlay,
              Float* __restrict__ var1_full, Float* __restrict__ var2_full, Float* __restrict__ var3_full,  Float* __restrict__ var4_full, Float* __restrict__ var5_full,
              const Float* __restrict__ var1_sub, const Float* __restrict__ var2_sub, const Float* __restrict__ var3_sub, const Float* __restrict__ var4_sub, const Float* __restrict__ var5_sub)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int ilay = blockIdx.y*blockDim.y + threadIdx.y;

    if ( (icol < ncol) && (ilay < nlay) )
    {
        const int idx = icol + ilay*ncol;
        var1_full[idx] += var1_sub[idx];
        var2_full[idx] += var2_sub[idx];
        var3_full[idx] += var3_sub[idx];
        var4_full[idx] += var4_sub[idx];
        var5_full[idx] += var5_sub[idx];
    }
}

__global__
void add_from_gpoint_kernel(const int ncol, const int nlay,
              Float* __restrict__ var1_full, Float* __restrict__ var2_full, Float* __restrict__ var3_full,  Float* __restrict__ var4_full,
              const Float* __restrict__ var1_sub, const Float* __restrict__ var2_sub, const Float* __restrict__ var3_sub, const Float* __restrict__ var4_sub)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int ilay = blockIdx.y*blockDim.y + threadIdx.y;

    if ( (icol < ncol) && (ilay < nlay) )
    {
        const int idx = icol + ilay*ncol;
        var1_full[idx] += var1_sub[idx];
        var2_full[idx] += var2_sub[idx];
        var3_full[idx] += var3_sub[idx];
        var4_full[idx] += var4_sub[idx];
    }
}

__global__
void add_from_gpoint_kernel(const int ncol, const int nlay,
              Float* __restrict__ var1_full, Float* __restrict__ var2_full,
              const Float* __restrict__ var1_sub, const Float* __restrict__ var2_sub)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int ilay = blockIdx.y*blockDim.y + threadIdx.y;

    if ( (icol < ncol) && (ilay < nlay) )
    {
        const int idx = icol + ilay*ncol;
        var1_full[idx] += var1_sub[idx];
        var2_full[idx] += var2_sub[idx];
    }
}

__global__
void add_from_gpoint_kernel(const int ncol, const int nlay,
              Float* __restrict__ var1_full, const Float* __restrict__ var1_sub)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int ilay = blockIdx.y*blockDim.y + threadIdx.y;

    if ( (icol < ncol) && (ilay < nlay) )
    {
        const int idx = icol + ilay*ncol;
        var1_full[idx] += var1_sub[idx];
    }
}

__global__
void add_from_gpoint_kernel(const int ncol, const int nlay,
              Float* __restrict__ var1_full, Float* __restrict__ var2_full, Float* __restrict__ var3_full,
              const Float* __restrict__ var1_sub, const Float* __restrict__ var2_sub, const Float* __restrict__ var3_sub)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int ilay = blockIdx.y*blockDim.y + threadIdx.y;

    if ( (icol < ncol) && (ilay < nlay) )
    {
        const int idx = icol + ilay*ncol;
        var1_full[idx] += var1_sub[idx];
        var2_full[idx] += var2_sub[idx];
        var3_full[idx] += var3_sub[idx];
    }
}

__global__
void get_from_gpoint_kernel(const int ncol, const int nlay, const int igpt,
              Float* __restrict__ var1_full, Float* __restrict__ var2_full, Float* __restrict__ var3_full,  Float* __restrict__ var4_full,
              const Float* __restrict__ var1_sub, const Float* __restrict__ var2_sub, const Float* __restrict__ var3_sub, const Float* __restrict__ var4_sub)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int ilay = blockIdx.y*blockDim.y + threadIdx.y;

    if ( (icol < ncol) && (ilay < nlay) )
    {
        const int idx_full = icol + ilay*ncol + igpt*nlay*ncol;
        const int idx_sub = icol + ilay*ncol;
        var1_full[idx_full] = var1_sub[idx_sub];
        var2_full[idx_full] = var2_sub[idx_sub];
        var3_full[idx_full] = var3_sub[idx_sub];
        var4_full[idx_full] = var4_sub[idx_sub];
    }
}

__global__
void get_from_gpoint_kernel(const int ncol, const int nlay, const int igpt,
              Float* __restrict__ var1_full, Float* __restrict__ var2_full, Float* __restrict__ var3_full,
              const Float* __restrict__ var1_sub, const Float* __restrict__ var2_sub, const Float* __restrict__ var3_sub)
{
    const int icol = blockIdx.x*blockDim.x + threadIdx.x;
    const int ilay = blockIdx.y*blockDim.y + threadIdx.y;

    if ( (icol < ncol) && (ilay < nlay ) )
    {
        const int idx_full = icol + ilay*ncol + igpt*nlay*ncol;
        const int idx_sub = icol + ilay*ncol;
        var1_full[idx_full] = var1_sub[idx_sub];
        var2_full[idx_full] = var2_sub[idx_sub];
        var3_full[idx_full] = var3_sub[idx_sub];
    }
}

#include "hip/hip_runtime.h"
/*
 * This file is part of a C++ interface to the Radiative Transfer for Energetics (RTE)
 * and Rapid Radiative Transfer Model for GCM applications Parallel (RRTMGP).
 *
 * The original code is found at https://github.com/earth-system-radiation/rte-rrtmgp.
 *
 * Contacts: Robert Pincus and Eli Mlawer
 * email: rrtmgp@aer.com
 *
 * Copyright 2015-2020,  Atmospheric and Environmental Research and
 * Regents of the University of Colorado.  All right reserved.
 *
 * This C++ interface can be downloaded from https://github.com/earth-system-radiation/rte-rrtmgp-cpp
 *
 * Contact: Chiel van Heerwaarden
 * email: chiel.vanheerwaarden@wur.nl
 *
 * Copyright 2020, Wageningen University & Research.
 *
 * Use and duplication is permitted under the terms of the
 * BSD 3-clause license, see http://opensource.org/licenses/BSD-3-Clause
 *
 */

#include "Optical_props.h"
#include "Array.h"


#include "optical_props_kernels_cuda.h"


// Optical properties per gpoint.
Optical_props_gpu::Optical_props_gpu(
        const Array<Float,2>& band_lims_wvn,
        const Array<int,2>& band_lims_gpt)
{
    Array<int,2> band_lims_gpt_lcl(band_lims_gpt);
    Array_gpu<int,2> band_lims_gpt_lcl_gpu(band_lims_gpt);

    this->band2gpt = band_lims_gpt_lcl;
    this->band2gpt_gpu = this->band2gpt;
    this->band_lims_wvn = band_lims_wvn;

    // Make a map between g-points and bands.
    this->gpt2band.set_dims({band_lims_gpt_lcl.max()});
    for (int iband=1; iband<=band_lims_gpt_lcl.dim(2); ++iband)
    {
        for (int i=band_lims_gpt_lcl({1,iband}); i<=band_lims_gpt_lcl({2,iband}); ++i)
            this->gpt2band({i}) =  iband;
    }
    this->gpt2band_gpu = this->gpt2band;
}


// Optical properties per band.
Optical_props_gpu::Optical_props_gpu(
        const Array<Float,2>& band_lims_wvn)
{
    Array<int,2> band_lims_gpt_lcl({2, band_lims_wvn.dim(2)});

    for (int iband=1; iband<=band_lims_wvn.dim(2); ++iband)
    {
        band_lims_gpt_lcl({1, iband}) = iband;
        band_lims_gpt_lcl({2, iband}) = iband;
    }

    this->band2gpt = band_lims_gpt_lcl;
    this->band2gpt_gpu = this->band2gpt;
    this->band_lims_wvn = band_lims_wvn;

    // Make a map between g-points and bands.
    this->gpt2band.set_dims({band_lims_gpt_lcl.max()});
    for (int iband=1; iband<=band_lims_gpt_lcl.dim(2); ++iband)
    {
        for (int i=band_lims_gpt_lcl({1,iband}); i<=band_lims_gpt_lcl({2,iband}); ++i)
            this->gpt2band({i}) =  iband;
    }
    this->gpt2band_gpu = this->gpt2band;
}


Optical_props_1scl_gpu::Optical_props_1scl_gpu(
        const int ncol,
        const int nlay,
        const Optical_props_gpu& optical_props_gpu) :
    Optical_props_arry_gpu(optical_props_gpu),
    tau({ncol, nlay, this->get_ngpt()})
{}



Optical_props_2str_gpu::Optical_props_2str_gpu(
        const int ncol,
        const int nlay,
        const Optical_props_gpu& optical_props_gpu) :
    Optical_props_arry_gpu(optical_props_gpu),
    tau({ncol, nlay, this->get_ngpt()}),
    ssa({ncol, nlay, this->get_ngpt()}),
    g  ({ncol, nlay, this->get_ngpt()})
{}



void Optical_props_2str_gpu::delta_scale(const Array_gpu<Float,3>& forward_frac)
{
    const int ncol = this->get_ncol();
    const int nlay = this->get_nlay();
    const int ngpt = this->get_ngpt();

    Optical_props_kernels_cuda::delta_scale_2str_k(
            ncol, nlay, ngpt,
            this->get_tau().ptr(), this->get_ssa().ptr(), this->get_g().ptr());
}


void add_to(Optical_props_1scl_gpu& op_inout, const Optical_props_1scl_gpu& op_in)
{
    const int ncol = op_inout.get_ncol();
    const int nlay = op_inout.get_nlay();
    const int ngpt = op_inout.get_ngpt();

    if (ngpt == op_in.get_ngpt())
    {
        Optical_props_kernels_cuda::increment_1scalar_by_1scalar(
                ncol, nlay, ngpt,
                op_inout.get_tau().ptr(), op_in.get_tau().ptr());
    }
    else
    {
        if (op_in.get_ngpt() != op_inout.get_nband())
            throw std::runtime_error("Cannot add optical properties with incompatible band - gpoint combination");

        Optical_props_kernels_cuda::inc_1scalar_by_1scalar_bybnd(
                ncol, nlay, ngpt,
                op_inout.get_tau().ptr(), op_in.get_tau().ptr(),
                op_inout.get_nband(), op_inout.get_band_lims_gpoint_gpu().ptr());
    }
}


void add_to(Optical_props_2str_gpu& op_inout, const Optical_props_2str_gpu& op_in)
{
    const int ncol = op_inout.get_ncol();
    const int nlay = op_inout.get_nlay();
    const int ngpt = op_inout.get_ngpt();

    if (ngpt == op_in.get_ngpt())
    {
        Optical_props_kernels_cuda::increment_2stream_by_2stream(
                ncol, nlay, ngpt,
                op_inout.get_tau().ptr(), op_inout.get_ssa().ptr(), op_inout.get_g().ptr(),
                op_in   .get_tau().ptr(), op_in   .get_ssa().ptr(), op_in   .get_g().ptr());
    }
    else
    {
        if (op_in.get_ngpt() != op_inout.get_nband())
            throw std::runtime_error("Cannot add optical properties with incompatible band - gpoint combination");

        Optical_props_kernels_cuda::inc_2stream_by_2stream_bybnd(
                ncol, nlay, ngpt,
                op_inout.get_tau().ptr(), op_inout.get_ssa().ptr(), op_inout.get_g().ptr(),
                op_in   .get_tau().ptr(), op_in   .get_ssa().ptr(), op_in   .get_g().ptr(),
                op_inout.get_nband(), op_inout.get_band_lims_gpoint_gpu().ptr());
    }
}

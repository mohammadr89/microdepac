#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <iostream>

#include "raytracer_kernels.h"
#include "raytracer_definitions.h"

namespace
{
    using namespace Raytracer_functions;

    constexpr Float w_thres = 0.5;
    //constexpr Float solar_cone_cos_half_angle = Float(0.9961947); // cos(Float(5.0) / Float(180.) * M_PI;)
    constexpr Float solar_cone_cos_half_angle = Float(0.99904823); // cos(Float(2.5) / Float(180.) * M_PI;)

    struct Quasi_random_number_generator_2d
    {
        __device__ Quasi_random_number_generator_2d(
                hiprandDirectionVectors32_t* vectors, unsigned int* constants, unsigned int offset)
        {
            hiprand_init(vectors[0], constants[0], offset, &state_x);
            hiprand_init(vectors[1], constants[1], offset, &state_y);
        }

        __device__ void xy(unsigned int* x, unsigned int* y,
                           const Vector<int>& grid_cells,
                           const Int qrng_grid_x, const Int qrng_grid_y,
                           Int& photons_shot)
        {
            *x = hiprand(&state_x);
            *y = hiprand(&state_y);

            while (true)
            {
                const int i = *x / static_cast<unsigned int>((1ULL << 32) / qrng_grid_x);
                const int j = *y / static_cast<unsigned int>((1ULL << 32) / qrng_grid_y);

                ++photons_shot;
                if (i < grid_cells.x && j < grid_cells.y)
                {
                    return;
                }
                else
                {
                    *x = hiprand(&state_x);
                    *y = hiprand(&state_y);
                }
            }
        }

        hiprandStateScrambledSobol32_t state_x;
        hiprandStateScrambledSobol32_t state_y;
    };

    __device__
    inline void reset_photon(
            Photon& photon, Int& photons_shot, const Int photons_to_shoot,
            const Int qrng_grid_x, const Int qrng_grid_y,
            Float* __restrict__ const toa_down_count,
            Quasi_random_number_generator_2d& qrng,
            Random_number_generator<Float>& rng,
            const Float tod_inc_direct, const Float tod_inc_diffuse,
            const Vector<Float> grid_size,
            const Vector<Float> grid_d,
            const Vector<int> grid_cells,
            const Vector<Float> sun_direction,
            Float& weight)
    {
        unsigned int random_number_x;
        unsigned int random_number_y;
        qrng.xy(&random_number_x, &random_number_y, grid_cells, qrng_grid_x, qrng_grid_y, photons_shot);

        if (photons_shot < photons_to_shoot)
        {
            const int i = random_number_x / static_cast<unsigned int>((1ULL << 32) / qrng_grid_x);
            const int j = random_number_y / static_cast<unsigned int>((1ULL << 32) / qrng_grid_y);

            photon.position.x = grid_size.x * random_number_x / static_cast<unsigned int>((1ULL << 32) / qrng_grid_x) / grid_cells.x;
            photon.position.y = grid_size.y * random_number_y / static_cast<unsigned int>((1ULL << 32) / qrng_grid_y) / grid_cells.y;
            photon.position.z = grid_size.z;

            const Float tod_diff_frac = tod_inc_diffuse / (tod_inc_direct + tod_inc_diffuse);
            if (rng() >= tod_diff_frac)
            {
                photon.direction = sun_direction;
                photon.kind = Photon_kind::Direct;
            }
            else
            {
                const Float mu_surface = sqrt(rng());
                const Float azimuth_surface = Float(2.*M_PI)*rng();

                photon.direction.x = mu_surface*sin(azimuth_surface);
                photon.direction.y = mu_surface*cos(azimuth_surface);
                photon.direction.z = Float(-1) * (sqrt(Float(1.) - mu_surface*mu_surface + Float_epsilon));
                photon.kind = Photon_kind::Diffuse;
            }

            const int ij = i + j*grid_cells.x;

            #ifndef NDEBUG
            if (ij < 0 || ij >=grid_cells.x*grid_cells.y) printf("Out of Bounds in reset photon \n");
            #endif

            atomicAdd(&toa_down_count[ij], Float(1.));
            weight = 1;

        }
    }

    template<typename T> __device__
    inline Float from_solar_cone(
        const Vector<T>& sun_direction,
        const Vector<T>& photon_dir)
    {
        return dot(sun_direction, photon_dir) > solar_cone_cos_half_angle;
    }
}


__global__
void ray_tracer_kernel(
        const Int photons_to_shoot,
        const Int qrng_grid_x,
        const Int qrng_grid_y,
        const Int qrng_gpt_offset,
        const Float* __restrict__ k_null_grid,
        Float* __restrict__ toa_down_count,
        Float* __restrict__ tod_up_count,
        Float* __restrict__ surface_down_direct_count,
        Float* __restrict__ surface_down_diffuse_count,
        Float* __restrict__ surface_up_count,
        Float* __restrict__ atmos_direct_count,
        Float* __restrict__ atmos_diffuse_count,
        const Float* __restrict__ k_ext,
        const Optics_scat* __restrict__ scat_asy,
        const Float* __restrict__ r_eff,
        const Float tod_inc_direct,
        const Float tod_inc_diffuse,
        const Float* __restrict__ surface_albedo,
        const Vector<Float> grid_size,
        const Vector<Float> grid_d,
        const Vector<int> grid_cells,
        const Vector<int> kn_grid,
        const Vector<Float> sun_direction,
        hiprandDirectionVectors32_t* qrng_vectors, unsigned int* qrng_constants,
        const Float* __restrict__ mie_cdf,
        const Float* __restrict__ mie_ang,
        const int mie_table_size)
{
    extern __shared__ Float mie_cdf_shared[];
    if (threadIdx.x==0 && mie_table_size > 0)
    {
        for (int mie_i=0; mie_i<mie_table_size; ++mie_i)
        {
            mie_cdf_shared[mie_i] = mie_cdf[mie_i];
        }
    }

    __syncthreads();

    const Vector<Float> kn_grid_d = grid_size / kn_grid;

    const int n = blockDim.x * blockIdx.x + threadIdx.x;

    Photon photon;
    Random_number_generator<Float> rng(n+qrng_gpt_offset);
    Quasi_random_number_generator_2d qrng(qrng_vectors, qrng_constants, n*photons_to_shoot + qrng_gpt_offset);

    const Float s_min = max(grid_size.z, max(grid_size.y, grid_size.x)) * Float_epsilon;

    // Set up the initial photons.
    Int photons_shot = Atomic_reduce_const;
    Float weight;

    reset_photon(
            photon, photons_shot, photons_to_shoot,
            qrng_grid_x, qrng_grid_y,
            toa_down_count,
            qrng, rng,
            tod_inc_direct, tod_inc_diffuse,
            grid_size, grid_d,
            grid_cells, sun_direction,
            weight);

    Float tau = Float(0.);
    Float d_max = Float(0.);
    Float k_ext_null;
    Bool transition = false;
    int i_n, j_n, k_n;

    while (photons_shot < photons_to_shoot)
    {
        // if d_max is zero, find current grid and maximum distance
        if (d_max == Float(0.))
        {
            i_n = float_to_int(photon.position.x, kn_grid_d.x, kn_grid.x);
            j_n = float_to_int(photon.position.y, kn_grid_d.y, kn_grid.y);
            k_n = float_to_int(photon.position.z, kn_grid_d.z, kn_grid.z);
            const Float sx = abs((photon.direction.x > 0) ? ((i_n+1) * kn_grid_d.x - photon.position.x)/photon.direction.x : (i_n*kn_grid_d.x - photon.position.x)/photon.direction.x);
            const Float sy = abs((photon.direction.y > 0) ? ((j_n+1) * kn_grid_d.y - photon.position.y)/photon.direction.y : (j_n*kn_grid_d.y - photon.position.y)/photon.direction.y);
            const Float sz = abs((photon.direction.z > 0) ? ((k_n+1) * kn_grid_d.z - photon.position.z)/photon.direction.z : (k_n*kn_grid_d.z - photon.position.z)/photon.direction.z);
            d_max = min(sx, min(sy, sz));
            const int ijk_n = i_n + j_n*kn_grid.x + k_n*kn_grid.x*kn_grid.y;
            k_ext_null = k_null_grid[ijk_n];
        }

        if (!transition)
        {
            tau = sample_tau(rng());
        }
        transition = false;
        const Float dn = max(Float_epsilon, tau / k_ext_null);

        if (dn >= d_max)
        {
            const Float dx = photon.direction.x * (s_min + d_max);
            const Float dy = photon.direction.y * (s_min + d_max);
            const Float dz = photon.direction.z * (s_min + d_max);

            photon.position.x += dx;
            photon.position.y += dy;
            photon.position.z += dz;

            // surface hit
            if (photon.position.z < Float_epsilon)
            {
                photon.position.z = Float_epsilon;
                const int i = float_to_int(photon.position.x, grid_d.x, grid_cells.x);
                const int j = float_to_int(photon.position.y, grid_d.y, grid_cells.y);
                const int ij = i + j*grid_cells.x;
                d_max = Float(0.);

                #ifndef NDEBUG
                if (ij < 0 || ij >=grid_cells.x*grid_cells.y)
                {
                    printf("outofbounds 1 \n");
                }
                #endif

                // // Add surface irradiance
                if (photon.kind == Photon_kind::Direct)
                    write_photon_out(&surface_down_direct_count[ij], weight);
                else if (photon.kind == Photon_kind::Diffuse)
                    write_photon_out(&surface_down_diffuse_count[ij], weight);
                // if (from_solar_cone(sun_direction, photon.direction))
                //     write_photon_out(&surface_down_direct_count[ij], weight);
                // else
                //     write_photon_out(&surface_down_diffuse_count[ij], weight);

                // Update weights and add upward surface flux
                const Float local_albedo = surface_albedo[0];
                weight *= local_albedo;
                write_photon_out(&surface_up_count[ij], weight);

                if (weight < w_thres)
                    weight = (rng() > weight) ? Float(0.) : Float(1.);

                // only with nonzero weight continue ray tracing, else start new ray
                if (weight > Float(0.))
                {
                    const Float mu_surface = sqrt(rng());
                    const Float azimuth_surface = Float(2.*M_PI)*rng();

                    photon.direction.x = mu_surface*sin(azimuth_surface);
                    photon.direction.y = mu_surface*cos(azimuth_surface);
                    photon.direction.z = sqrt(Float(1.) - mu_surface*mu_surface + Float_epsilon);
                    photon.kind = Photon_kind::Diffuse;
                }
                else
                {
                    reset_photon(
                            photon, photons_shot, photons_to_shoot,
                            qrng_grid_x, qrng_grid_y,
                            toa_down_count,
                            qrng, rng,
                            tod_inc_direct, tod_inc_diffuse,
                            grid_size, grid_d,
                            grid_cells, sun_direction,
                            weight);
                }
            }

            // TOD exit
            else if (photon.position.z >= grid_size.z)
            {
                d_max = Float(0.);

                const int i = float_to_int(photon.position.x, grid_d.x, grid_cells.x);
                const int j = float_to_int(photon.position.y, grid_d.y, grid_cells.y);
                const int ij = i + j*grid_cells.x;

                #ifndef NDEBUG
                if (ij < 0 || ij >=grid_cells.x*grid_cells.y) printf("Out of bounds at TOD \n");
                #endif

                write_photon_out(&tod_up_count[ij], weight);

                reset_photon(
                        photon, photons_shot, photons_to_shoot,
                        qrng_grid_x, qrng_grid_y,
                        toa_down_count,
                        qrng, rng,
                        tod_inc_direct, tod_inc_diffuse,
                        grid_size, grid_d,
                        grid_cells, sun_direction,
                        weight);

            }
            // regular cell crossing: adjust tau and apply periodic BC
            else
            {
                photon.position.x += photon.direction.x>0 ? s_min : -s_min;
                photon.position.y += photon.direction.y>0 ? s_min : -s_min;
                photon.position.z += photon.direction.z>0 ? s_min : -s_min;

                // Cyclic boundary condition in x.
                photon.position.x = fmod(photon.position.x, grid_size.x);
                if (photon.position.x < Float(0.))
                    photon.position.x += grid_size.x;

                // Cyclic boundary condition in y.
                photon.position.y = fmod(photon.position.y, grid_size.y);
                if (photon.position.y < Float(0.))
                    photon.position.y += grid_size.y;

                tau -= d_max * k_ext_null;
                d_max = Float(0.);
                transition = true;
            }
        }
        else
        {
            Float dx = photon.direction.x * dn;
            Float dy = photon.direction.y * dn;
            Float dz = photon.direction.z * dn;

            photon.position.x = (dx > 0) ? min(photon.position.x + dx, (i_n+1) * kn_grid_d.x - s_min) : max(photon.position.x + dx, (i_n) * kn_grid_d.x + s_min);
            photon.position.y = (dy > 0) ? min(photon.position.y + dy, (j_n+1) * kn_grid_d.y - s_min) : max(photon.position.y + dy, (j_n) * kn_grid_d.y + s_min);
            photon.position.z = (dz > 0) ? min(photon.position.z + dz, (k_n+1) * kn_grid_d.z - s_min) : max(photon.position.z + dz, (k_n) * kn_grid_d.z + s_min);

            // Calculate the 3D index.
            const int i = float_to_int(photon.position.x, grid_d.x, grid_cells.x);
            const int j = float_to_int(photon.position.y, grid_d.y, grid_cells.y);
            const int k = float_to_int(photon.position.z, grid_d.z, grid_cells.z);
            const int ijk = i + j*grid_cells.x + k*grid_cells.x*grid_cells.y;

            // Compute probability not being absorbed and store weighted absorption probability
            const Float k_sca_tot = scat_asy[ijk].k_sca_gas + scat_asy[ijk].k_sca_cld + scat_asy[ijk].k_sca_aer;
            const Float ssa_tot = k_sca_tot / k_ext[ijk];

            const Float f_no_abs = Float(1.) - (Float(1.) - ssa_tot) * (k_ext[ijk]/k_ext_null);

            #ifndef NDEBUG
            if (ijk < 0 || ijk >= grid_cells.x*grid_cells.y*grid_cells.z) printf("Out of Bounds at Heating Rates \n");
            #endif

            if (photon.kind == Photon_kind::Direct)
                write_photon_out(&atmos_direct_count[ijk], weight*(1-f_no_abs));
            else
                write_photon_out(&atmos_diffuse_count[ijk], weight*(1-f_no_abs));

            // Update weights (see Iwabuchi 2006: https://doi.org/10.1175/JAS3755.1)
            weight *= f_no_abs;
            if (weight < w_thres)
                weight = (rng() > weight) ? Float(0.) : Float(1.);

            // only with nonzero weight continue ray tracing, else start new ray
            if (weight > Float(0.))
            {
                // Null collision.
                if (rng() >= ssa_tot / (ssa_tot - Float(1.) + k_ext_null / k_ext[ijk]))
                {
                    d_max -= dn;
                }
                // Scattering.
                else
                {
                    d_max = Float(0.);
                    // find scatter type: 0 = gas, 1 = cloud, 2 = aerosol
                    const Float scatter_rng = rng();
                    const int scatter_type = scatter_rng < (scat_asy[ijk].k_sca_aer/k_sca_tot) ? 2 :
                                             scatter_rng < ((scat_asy[ijk].k_sca_aer+scat_asy[ijk].k_sca_cld)/k_sca_tot) ? 1 : 0;
                    Float g;
                    switch (scatter_type)
                    {
                        case 0:
                            g = Float(0.);
                            break;
                        case 1:
                            g = min(Float(1.) - Float_epsilon, scat_asy[ijk].asy_cld);
                            break;
                        case 2:
                            g = min(Float(1.) - Float_epsilon, scat_asy[ijk].asy_aer);
                            break;
                    }

                    // 0 (gas): rayleigh, 1 (cloud): mie if mie_table_size>0 else HG, 2 (aerosols) HG
                    const Float cos_scat = scatter_type == 0 ? rayleigh(rng()) : // gases -> rayleigh,
                                                           1 ? ( (mie_table_size > 0) //clouds: Mie or HG
                                                                    ? cos( mie_sample_angle(mie_cdf_shared, mie_ang, rng(), r_eff[ijk], mie_table_size) )
                                                                    :  henyey(g, rng()))
                                                           : henyey(g, rng()); //aerosols
                    const Float sin_scat = max(Float(0.), sqrt(Float(1.) - cos_scat*cos_scat + Float_epsilon));

                    Vector<Float> t1{Float(0.), Float(0.), Float(0.)};
                    if (fabs(photon.direction.x) < fabs(photon.direction.y))
                    {
                        if (fabs(photon.direction.x) < fabs(photon.direction.z))
                            t1.x = Float(1.);
                        else
                            t1.z = Float(1.);
                    }
                    else
                    {
                        if (fabs(photon.direction.y) < fabs(photon.direction.z))
                            t1.y = Float(1.);
                        else
                            t1.z = Float(1.);
                    }
                    t1 = normalize(t1 - photon.direction*dot(t1, photon.direction));
                    Vector<Float> t2 = cross(photon.direction, t1);

                    const Float phi = Float(2.*M_PI)*rng();

                    photon.direction = cos_scat*photon.direction
                            + sin_scat*(sin(phi)*t1 + cos(phi)*t2);

                    photon.kind = Photon_kind::Diffuse;
                }
            }
            else
            {
                d_max = Float(0.);
                reset_photon(
                        photon, photons_shot, photons_to_shoot,
                        qrng_grid_x, qrng_grid_y,
                        toa_down_count,
                        qrng, rng,
                        tod_inc_direct, tod_inc_diffuse,
                        grid_size, grid_d,
                        grid_cells, sun_direction,
                        weight);
            }
        }
    }
}

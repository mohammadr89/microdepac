#include "hip/hip_runtime.h"
#include <chrono>
#include <functional>
#include <iostream>
#include <iomanip>

#include "gas_optics_rrtmgp_kernels_cuda_rt.h"
#include "tools_gpu.h"
#include "tuner.h"


namespace
{
    #include "gas_optics_rrtmgp_kernels_rt.cu"

    using Tools_gpu::calc_grid_size;
}


namespace Gas_optics_rrtmgp_kernels_cuda_rt
{
    void reorder123x321(
            const int ni, const int nj, const int nk,
            const Float* arr_in, Float* arr_out)
    {
        Tuner_map& tunings = Tuner::get_map();

        dim3 grid(ni, nj, nk);
        dim3 block;

        if (tunings.count("reorder123x321_kernel_rt") == 0)
        {
            std::tie(grid, block) = tune_kernel(
                "reorder123x321_kernel_rt",
                dim3(ni, nj, nk),
                {1, 2, 4, 8, 16, 24, 32, 48, 64, 96},
                {1, 2, 4, 8, 16, 24, 32, 48, 64, 96},
                {1, 2, 4, 8, 16, 24, 32, 48, 64, 96},
                reorder123x321_kernel,
                ni, nj, nk, arr_in, arr_out);

            tunings["reorder123x321_kernel_rt"].first = grid;
            tunings["reorder123x321_kernel_rt"].second = block;
        }
        else
        {
            block = tunings["reorder123x321_kernel_rt"].second;
        }

        grid = calc_grid_size(block, dim3(ni, nj, nk));

        reorder123x321_kernel<<<grid, block>>>(
                ni, nj, nk, arr_in, arr_out);
    }


    void reorder12x21(const int ni, const int nj,
                      const Float* arr_in, Float* arr_out)
    {
        const int block_i = 32;
        const int block_j = 16;

        const int grid_i = ni/block_i + (ni%block_i > 0);
        const int grid_j = nj/block_j + (nj%block_j > 0);

        dim3 grid_gpu(grid_i, grid_j);
        dim3 block_gpu(block_i, block_j);

        reorder12x21_kernel<<<grid_gpu, block_gpu>>>(
                ni, nj, arr_in, arr_out);
    }


    void zero_array(const int ni, const int nj, const int nk, const int nn, Float* arr)
    {
        const int block_i = 32;
        const int block_j = 16;
        const int block_k = 1;

        const int grid_i = ni/block_i + (ni%block_i > 0);
        const int grid_j = nj/block_j + (nj%block_j > 0);
        const int grid_k = nk/block_k + (nk%block_k > 0);

        dim3 grid_gpu(grid_i, grid_j, grid_k);
        dim3 block_gpu(block_i, block_j, block_k);

        zero_array_kernel<<<grid_gpu, block_gpu>>>(
                ni, nj, nk, nn, arr);
    }


    void zero_array(const int ni, const int nj, const int nk, Float* arr)
    {
        const int block_i = 32;
        const int block_j = 16;
        const int block_k = 1;

        const int grid_i = ni/block_i + (ni%block_i > 0);
        const int grid_j = nj/block_j + (nj%block_j > 0);
        const int grid_k = nk/block_k + (nk%block_k > 0);

        dim3 grid_gpu(grid_i, grid_j, grid_k);
        dim3 block_gpu(block_i, block_j, block_k);

        zero_array_kernel<<<grid_gpu, block_gpu>>>(
                ni, nj, nk, arr);

    }


    void zero_array(const int ni, const int nj, Float* arr)
    {
        const int block_i = 32;
        const int block_j = 16;

        const int grid_i = ni/block_i + (ni%block_i > 0);
        const int grid_j = nj/block_j + (nj%block_j > 0);

        dim3 grid_gpu(grid_i, grid_j, 1);
        dim3 block_gpu(block_i, block_j, 1);

        zero_array_kernel<<<grid_gpu, block_gpu>>>(
                ni, nj, arr);

    }

    void zero_array(const int ni, int* arr)
    {
        const int block_i = 32;

        const int grid_i = ni/block_i + (ni%block_i > 0);

        dim3 grid_gpu(grid_i);
        dim3 block_gpu(block_i);

        zero_array_kernel<<<grid_gpu, block_gpu>>>(
                ni, arr);
    }

    void interpolation(
            const int col_s, const int ncol_sub, const int ncol, const int nlay, const int igpt,
            const int ngas, const int nflav, const int neta, const int npres, const int ntemp,
            const int* gpoint_flavor,
            const int* flavor,
            const Float* press_ref_log,
            const Float* temp_ref,
            Float press_ref_log_delta,
            Float temp_ref_min,
            Float temp_ref_delta,
            Float press_ref_trop_log,
            const Float* vmr_ref,
            const Float* play,
            const Float* tlay,
            Float* col_gas,
            int* jtemp,
            Float* fmajor, Float* fminor,
            Float* col_mix,
            Bool* tropo,
            int* jeta,
            int* jpress)
    {
        Tuner_map& tunings = Tuner::get_map();
        Float tmin = std::numeric_limits<Float>::min();

        dim3 grid(nlay, ncol_sub, 1), block;
        if (tunings.count("interpolation_kernel_rt") == 0)
        {
            std::tie(grid, block) = tune_kernel(
                    "interpolation_kernel_rt",
                    dim3(nlay, ncol_sub, 1),
                    {1,2,4}, {1, 2, 4, 8, 16, 32, 64, 128, 256, 512}, {1},
                    interpolation_kernel,
                    igpt-1, col_s, ncol_sub, ncol, nlay, ngas, nflav, neta, npres, ntemp, tmin,
                    gpoint_flavor, flavor, press_ref_log, temp_ref,
                    press_ref_log_delta, temp_ref_min,
                    temp_ref_delta, press_ref_trop_log,
                    vmr_ref, play, tlay,
                    col_gas, jtemp, fmajor,
                    fminor, col_mix, tropo,
                    jeta, jpress);
            tunings["interpolation_kernel_rt"].first = grid;
            tunings["interpolation_kernel_rt"].second = block;
        }
        else
        {
            block = tunings["interpolation_kernel_rt"].second;
        }

        grid = calc_grid_size(block, dim3(nlay, ncol_sub, 1));

        interpolation_kernel<<<grid, block>>>(
                igpt-1, col_s, ncol_sub, ncol, nlay, ngas, nflav, neta, npres, ntemp, tmin,
                gpoint_flavor, flavor, press_ref_log, temp_ref,
                press_ref_log_delta, temp_ref_min,
                temp_ref_delta, press_ref_trop_log,
                vmr_ref, play, tlay,
                col_gas, jtemp, fmajor,
                fminor, col_mix, tropo,
                jeta, jpress);

    }

    void combine_abs_and_rayleigh(
            const int col_s, const int ncol_sub, const int ncol, const int nlay,
            const Float* tau_abs, const Float* tau_rayleigh,
            Float* tau, Float* ssa, Float* g)
    {
        Tuner_map& tunings = Tuner::get_map();

        Float tmin = std::numeric_limits<Float>::epsilon();

        dim3 grid(ncol_sub, nlay, 1);
        dim3 block;

        if (tunings.count("combine_abs_and_rayleigh_kernel_rt") == 0)
        {
            std::tie(grid, block) = tune_kernel(
                "combine_abs_and_rayleigh_kernel_rt",
                dim3(ncol_sub, nlay, 1),
                {24, 32, 48, 64, 96, 128, 256, 512}, {1, 2, 4}, {1},
                combine_abs_and_rayleigh_kernel,
                col_s, ncol_sub, ncol, nlay, tmin,
                tau_abs, tau_rayleigh,
                tau, ssa, g);

            tunings["combine_abs_and_rayleigh_kernel_rt"].first = grid;
            tunings["combine_abs_and_rayleigh_kernel_rt"].second = block;
        }
        else
        {
            block = tunings["combine_abs_and_rayleigh_kernel_rt"].second;
        }

        grid = calc_grid_size(block, dim3(ncol_sub, nlay, 1));

        combine_abs_and_rayleigh_kernel<<<grid, block>>>(
                col_s, ncol_sub, ncol, nlay, tmin,
                tau_abs, tau_rayleigh,
                tau, ssa, g);
    }


    void compute_tau_rayleigh(
            const int col_s, const int ncol_sub, const int ncol, const int nlay, const int nbnd, const int ngpt, 
            const int igpt, const int ngas, const int nflav, const int neta, const int npres, const int ntemp,
            const int* gpoint_bands,
            const int* band_lims_gpt,
            const Float* krayl,
            int idx_h2o, const Float* col_dry, const Float* col_gas,
            const Float* fminor, const int* jeta,
            const Bool* tropo, const int* jtemp,
            Float* tau_rayleigh)
    {
        Tuner_map& tunings = Tuner::get_map();

        dim3 grid(ncol_sub, nlay, 1), block;
        if (tunings.count("compute_tau_rayleigh_kernel_rt") == 0)
        {
            std::tie(grid, block) = tune_kernel(
                "compute_tau_rayleigh_kernel_rt",
                dim3(ncol_sub, nlay, 1),
                {24, 32, 64, 128, 256, 512}, {1, 2}, {1},
                compute_tau_rayleigh_kernel,
                col_s, ncol_sub, ncol, nlay, nbnd, ngpt,
                ngas, nflav, neta, npres, ntemp,
                igpt-1,
                gpoint_bands,
                band_lims_gpt,
                krayl,
                idx_h2o, col_dry, col_gas,
                fminor, jeta,
                tropo, jtemp,
                tau_rayleigh);

            tunings["compute_tau_rayleigh_kernel_rt"].first = grid;
            tunings["compute_tau_rayleigh_kernel_rt"].second = block;
        }
        else
        {
            block = tunings["compute_tau_rayleigh_kernel_rt"].second;
        }

        grid = calc_grid_size(block, dim3(ncol_sub, nlay, 1));

        compute_tau_rayleigh_kernel<<<grid, block>>>(
                col_s, ncol_sub, ncol, nlay, nbnd, ngpt,
                ngas, nflav, neta, npres, ntemp,
                igpt-1,
                gpoint_bands,
                band_lims_gpt,
                krayl,
                idx_h2o, col_dry, col_gas,
                fminor, jeta,
                tropo, jtemp,
                tau_rayleigh);
    }


    void compute_tau_absorption(
            const int col_s, const int ncol_sub, const int ncol, const int nlay, const int nband, const int ngpt, 
            const int igpt, const int ngas, const int nflav, const int neta, const int npres, const int ntemp,
            const int nminorlower, const int nminorklower,
            const int nminorupper, const int nminorkupper,
            const int idx_h2o,
            const int* band_lims_gpt,
            const Float* kmajor,
            const Float* kminor_lower,
            const Float* kminor_upper,
            const int* minor_limits_gpt_lower,
            const int* minor_limits_gpt_upper,
            const int* first_last_minor_lower,
            const int* first_last_minor_upper,
            const Bool* minor_scales_with_density_lower,
            const Bool* minor_scales_with_density_upper,
            const Bool* scale_by_complement_lower,
            const Bool* scale_by_complement_upper,
            const int* idx_minor_lower,
            const int* idx_minor_upper,
            const int* idx_minor_scaling_lower,
            const int* idx_minor_scaling_upper,
            const int* kminor_start_lower,
            const int* kminor_start_upper,
            const Bool* tropo,
            const Float* col_mix, const Float* fmajor,
            const Float* fminor, const Float* play,
            const Float* tlay, const Float* col_gas,
            const int* jeta, const int* jtemp,
            const int* jpress,
            Float* tau)
    {
        Tuner_map& tunings = Tuner::get_map();

        dim3 grid_maj(nlay, ncol_sub, 1);
        dim3 block_maj;

        if (tunings.count("gas_optical_depths_major_kernel_rt") == 0)
        {
            Float* tau_tmp = Tools_gpu::allocate_gpu<Float>(nlay*ncol_sub);

            std::tie(grid_maj, block_maj) = tune_kernel(
                    "gas_optical_depths_major_kernel_rt",
                    dim3(nlay, ncol_sub, 1),
                    {1, 2}, {64, 96, 128, 256, 512, 768, 1024}, {1},
                    gas_optical_depths_major_kernel,
                    col_s, ncol_sub, ncol, nlay, nband, ngpt,
                    nflav, neta, npres, ntemp,
                    igpt-1, band_lims_gpt,
                    kmajor, col_mix, fmajor, jeta,
                    tropo, jtemp, jpress,
                    tau_tmp);

            Tools_gpu::free_gpu<Float>(tau_tmp);

            tunings["gas_optical_depths_major_kernel_rt"].first = grid_maj;
            tunings["gas_optical_depths_major_kernel_rt"].second = block_maj;
        }
        else
        {
            block_maj = tunings["gas_optical_depths_major_kernel_rt"].second;
        }

        grid_maj = calc_grid_size(block_maj, dim3(nlay, ncol_sub, 1));

        gas_optical_depths_major_kernel<<<grid_maj, block_maj>>>(
            col_s, ncol_sub, ncol, nlay, nband, ngpt,
            nflav, neta, npres, ntemp,
            igpt-1, band_lims_gpt,
            kmajor, col_mix, fmajor, jeta,
            tropo, jtemp, jpress,
            tau);


        const int nscale_lower = nminorlower;
        const int nscale_upper = nminorupper;

        // Lower
        int idx_tropo = 1;

        dim3 grid_min_1(nlay, ncol_sub, 1), block_min_1;
        if (tunings.count("gas_optical_depths_minor_kernel_lower_rt") == 0)
        {
            Float* tau_tmp = Tools_gpu::allocate_gpu<Float>(nlay*ncol_sub);

            std::tie(grid_min_1, block_min_1) = tune_kernel(
                        "gas_optical_depths_minor_kernel_lower_rt",
                        dim3(nlay, ncol_sub, 1),
                        {1}, {32, 48, 64, 96, 128, 256, 384, 512}, {1},
                        gas_optical_depths_minor_kernel,
                        col_s, ncol_sub, ncol, nlay, ngpt, igpt-1,
                        ngas, nflav, ntemp, neta,
                        nscale_lower,
                        nminorlower,
                        nminorklower,
                        idx_h2o, idx_tropo,
                        kminor_lower,
                        minor_limits_gpt_lower,
                        first_last_minor_lower,
                        minor_scales_with_density_lower,
                        scale_by_complement_lower,
                        idx_minor_lower,
                        idx_minor_scaling_lower,
                        kminor_start_lower,
                        play, tlay, col_gas,
                        fminor, jeta, jtemp,
                        tropo,
                        tau_tmp);
            Tools_gpu::free_gpu<Float>(tau_tmp);

            tunings["gas_optical_depths_minor_kernel_lower_rt"].first = grid_min_1;
            tunings["gas_optical_depths_minor_kernel_lower_rt"].second = block_min_1;
        }
        else
        {
            block_min_1 = tunings["gas_optical_depths_minor_kernel_lower_rt"].second;
        }
        
        grid_min_1 = calc_grid_size(block_min_1, dim3(nlay, ncol_sub, 1));

        gas_optical_depths_minor_kernel<<<grid_min_1, block_min_1>>>(
                col_s, ncol_sub, ncol, nlay, ngpt, igpt-1,
                ngas, nflav, ntemp, neta,
                nscale_lower,
                nminorlower,
                nminorklower,
                idx_h2o, idx_tropo,
                kminor_lower,
                minor_limits_gpt_lower,
                first_last_minor_lower,
                minor_scales_with_density_lower,
                scale_by_complement_lower,
                idx_minor_lower,
                idx_minor_scaling_lower,
                kminor_start_lower,
                play, tlay, col_gas,
                fminor, jeta, jtemp,
                tropo, tau);

        // Upper
        idx_tropo = 0;

        dim3 grid_min_2(nlay, ncol_sub, 1), block_min_2;
        if (tunings.count("gas_optical_depths_minor_kernel_upper_rt") == 0)
        {
            Float* tau_tmp = Tools_gpu::allocate_gpu<Float>(nlay*ncol_sub);
            std::tie(grid_min_2, block_min_2) = tune_kernel(
                   "gas_optical_depths_minor_kernel_upper_rt",
                   dim3(nlay, ncol_sub, 1),
                   {1}, {32, 48, 64, 96, 128, 256, 384, 512}, {1},
                   gas_optical_depths_minor_kernel,
                   col_s, ncol_sub, ncol, nlay, ngpt, igpt-1,
                   ngas, nflav, ntemp, neta,
                   nscale_upper,
                   nminorupper,
                   nminorkupper,
                   idx_h2o, idx_tropo,
                   kminor_upper,
                   minor_limits_gpt_upper,
                   first_last_minor_upper,
                   minor_scales_with_density_upper,
                   scale_by_complement_upper,
                   idx_minor_upper,
                   idx_minor_scaling_upper,
                   kminor_start_upper,
                   play, tlay, col_gas,
                   fminor, jeta, jtemp,
                   tropo,
                   tau_tmp);
            Tools_gpu::free_gpu<Float>(tau_tmp);

            tunings["gas_optical_depths_minor_kernel_upper_rt"].first = grid_min_2;
            tunings["gas_optical_depths_minor_kernel_upper_rt"].second = block_min_2;
        }
        else
        {
            block_min_2 = tunings["gas_optical_depths_minor_kernel_upper_rt"].second;
        }
        
        grid_min_2 = calc_grid_size(block_min_2, dim3(nlay, ncol_sub, 1));

        gas_optical_depths_minor_kernel<<<grid_min_2, block_min_2>>>(
                col_s, ncol_sub, ncol, nlay, ngpt, igpt-1,
                ngas, nflav, ntemp, neta,
                nscale_upper,
                nminorupper,
                nminorkupper,
                idx_h2o, idx_tropo,
                kminor_upper,
                minor_limits_gpt_upper,
                first_last_minor_upper,
                minor_scales_with_density_upper,
                scale_by_complement_upper,
                idx_minor_upper,
                idx_minor_scaling_upper,
                kminor_start_upper,
                play, tlay, col_gas,
                fminor, jeta, jtemp,
                tropo, tau);

    }



    void Planck_source(
            const int ncol, const int nlay, const int nbnd, const int ngpt, const int igpt,
            const int nflav, const int neta, const int npres, const int ntemp,
            const int nPlanckTemp,
            const Float* tlay,
            const Float* tlev,
            const Float* tsfc,
            const int sfc_lay,
            const Float* fmajor,
            const int* jeta,
            const Bool* tropo,
            const int* jtemp,
            const int* jpress,
            const int* gpoint_bands,
            const int* band_lims_gpt,
            const Float* pfracin,
            const Float temp_ref_min, const Float totplnk_delta,
            const Float* totplnk,
            Float* sfc_src,
            Float* lay_src,
            Float* lev_src_inc,
            Float* lev_src_dec,
            Float* sfc_src_jac)
    {
        Tuner_map& tunings = Tuner::get_map();

        const Float delta_Tsurf = Float(1.);

        dim3 grid(ncol, nlay, 1), block;
        if (tunings.count("Planck_source_kernel_rt") == 0)
        {
            std::tie(grid, block) = tune_kernel(
                    "Planck_source_kernel_rt",
                    dim3(ncol, nlay, 1),
                    {16, 32, 48, 64, 96, 128, 256, 512}, {1, 2, 4, 8}, {1},
                    Planck_source_kernel,
                    ncol, nlay, nbnd, ngpt,
                    nflav, neta, npres, ntemp, nPlanckTemp, igpt-1,
                    tlay, tlev, tsfc, sfc_lay,
                    fmajor, jeta, tropo, jtemp,
                    jpress, gpoint_bands, band_lims_gpt,
                    pfracin, temp_ref_min, totplnk_delta,
                    totplnk,
                    delta_Tsurf, sfc_src, lay_src,
                    lev_src_inc, lev_src_dec,
                    sfc_src_jac);

            tunings["Planck_source_kernel_rt"].first = grid;
            tunings["Planck_source_kernel_rt"].second = block;
        }
        else
        {
            block = tunings["Planck_source_kernel_rt"].second;
        }
        
        grid = calc_grid_size(block, dim3(ncol, nlay, 1));

        Planck_source_kernel<<<grid, block>>>(
                ncol, nlay, nbnd, ngpt,
                nflav, neta, npres, ntemp, nPlanckTemp, igpt-1,
                tlay, tlev, tsfc, sfc_lay,
                fmajor, jeta, tropo, jtemp,
                jpress, gpoint_bands, band_lims_gpt,
                pfracin, temp_ref_min, totplnk_delta,
                totplnk,
                delta_Tsurf,
                sfc_src, lay_src,
                lev_src_inc, lev_src_dec,
                sfc_src_jac);
    }
}


#include "hip/hip_runtime.h"
/*
 * This file is part of a C++ interface to the Radiative Transfer for Energetics (RTE)
 * and Rapid Radiative Transfer Model for GCM applications Parallel (RRTMGP).
 *
 * The original code is found at https://github.com/earth-system-radiation/rte-rrtmgp.
 *
 * Contacts: Robert Pincus and Eli Mlawer
 * email: rrtmgp@aer.com
 *
 * Copyright 2015-2020,  Atmospheric and Environmental Research and
 * Regents of the University of Colorado.  All right reserved.
 *
 * This C++ interface can be downloaded from https://github.com/earth-system-radiation/rte-rrtmgp-cpp
 *
 * Contact: Chiel van Heerwaarden
 * email: chiel.vanheerwaarden@wur.nl
 *
 * Copyright 2020, Wageningen University & Research.
 *
 * Use and duplication is permitted under the terms of the
 * BSD 3-clause license, see http://opensource.org/licenses/BSD-3-Clause
 *
 */

#include "Fluxes_rt.h"
#include "Array.h"
#include "Optical_props_rt.h"

namespace
{
    __global__
    void sum_broadband(
                const int ncol, const int nlev, const int ngpt,
                const Float* __restrict__ spectral_flux, Float* __restrict__ broadband_flux)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        const int ilev = blockIdx.y*blockDim.y + threadIdx.y;

        if ( ( icol < ncol) && (ilev < nlev) )
        {
            const int idx_out = icol + ilev*ncol;
            Float bb_flux_s = 0;
            for (int igpt=0; igpt < ngpt; ++igpt)
            {
                const int idx_in = icol + ilev*ncol + igpt*nlev*ncol;
                bb_flux_s += spectral_flux[idx_in];
            }
            broadband_flux[idx_out] = bb_flux_s;
        }
    }

    __global__
    void net_broadband_precalc(
                const int ncol, const int nlev,
                const Float* __restrict__ flux_dn, const Float* __restrict__ flux_up,
                Float* __restrict__ broadband_flux_net)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        const int ilev = blockIdx.y*blockDim.y + threadIdx.y;

        if ( ( icol < ncol) && (ilev < nlev) )
        {
            const int idx = icol + ilev*ncol;
            broadband_flux_net[idx] = flux_dn[idx] - flux_up[idx];
        }
    }

    __global__
    void sum_byband(
                const int ncol, const int nlev, const int ngpt, const int nbnd,
                const int* __restrict__ band_lims, const Float* __restrict__ spectral_flux,
                Float* __restrict__ byband_flux)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        const int ilev = blockIdx.y*blockDim.y + threadIdx.y;
        const int ibnd = blockIdx.z*blockDim.z + threadIdx.z;

        if ( ( icol < ncol) && (ilev < nlev) && (ibnd < nbnd) )
        {
            const int idx_bnd = icol + ilev*ncol + ibnd*ncol*nlev;
            const int gpt_start = band_lims[2*ibnd];
            const int gpt_end = band_lims[2*ibnd+1];

            byband_flux[idx_bnd] = 0;

            for (int igpt = gpt_start; igpt < gpt_end; ++igpt)
            {
                const int idx_gpt = icol + ilev*ncol + igpt*ncol*nlev;
                byband_flux[idx_bnd] += spectral_flux[idx_gpt];
            }
        }
    }

    __global__
    void net_byband_full(
                const int ncol, const int nlev, const int ngpt, const int nbnd,
                const int* __restrict__ band_lims, const Float* __restrict__ spectral_flux_dn,
                const Float* __restrict__ spectral_flux_up, Float* __restrict__ byband_flux_net)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        const int ilev = blockIdx.y*blockDim.y + threadIdx.y;
        const int ibnd = blockIdx.z*blockDim.z + threadIdx.z;

        if ( ( icol < ncol) && (ilev < nlev) && (ibnd < nbnd) )
        {
            const int idx_bnd = icol + ilev*ncol + ibnd*ncol*nlev;
            const int gpt_start = band_lims[2*ibnd];
            const int gpt_end = band_lims[2*ibnd+1];
            byband_flux_net[idx_bnd] = 0;
            for (int igpt = gpt_start; igpt < gpt_end; ++igpt)
            {
                const int idx_gpt = icol + ilev*ncol + igpt*ncol*nlev;
                byband_flux_net[idx_bnd] += spectral_flux_dn[idx_gpt] - spectral_flux_up[idx_gpt];
            }
        }
    }
}

//namespace rrtmgp_kernel_launcher
//{
//    
//    void sum_broadband(
//            int ncol, int nlev, int ngpt,
//            const Array<Float,3>& spectral_flux, Array<Float,2>& broadband_flux)
//    {
//        rrtmgp_kernels::sum_broadband(
//                &ncol, &nlev, &ngpt,
//                const_cast<Float*>(spectral_flux.ptr()),
//                broadband_flux.ptr());
//    }
//
//    
//    void net_broadband(
//            int ncol, int nlev,
//            const Array<Float,2>& broadband_flux_dn, const Array<Float,2>& broadband_flux_up,
//            Array<Float,2>& broadband_flux_net)
//    {
//        rrtmgp_kernels::net_broadband_precalc(
//                &ncol, &nlev,
//                const_cast<Float*>(broadband_flux_dn.ptr()),
//                const_cast<Float*>(broadband_flux_up.ptr()),
//                broadband_flux_net.ptr());
//    }
//
//    
//    void sum_byband(
//            int ncol, int nlev, int ngpt, int nbnd,
//            const Array<int,2>& band_lims,
//            const Array<Float,3>& spectral_flux,
//            Array<Float,3>& byband_flux)
//    {
//        rrtmgp_kernels::sum_byband(
//                &ncol, &nlev, &ngpt, &nbnd,
//                const_cast<int*>(band_lims.ptr()),
//                const_cast<Float*>(spectral_flux.ptr()),
//                byband_flux.ptr());
//    }
//
//    
//    void net_byband(
//            int ncol, int nlev, int nband,
//            const Array<Float,3>& byband_flux_dn, const Array<Float,3>& byband_flux_up,
//            Array<Float,3>& byband_flux_net)
//    {
//        rrtmgp_kernels::net_byband_precalc(
//                &ncol, &nlev, &nband,
//                const_cast<Float*>(byband_flux_dn.ptr()),
//                const_cast<Float*>(byband_flux_up.ptr()),
//                byband_flux_net.ptr());
//    }


Fluxes_broadband_rt::Fluxes_broadband_rt(const int ncol_x, const int ncol_y, const int nlev) :
    flux_up     ({ncol_x*ncol_y, nlev}),
    flux_dn     ({ncol_x*ncol_y, nlev}),
    flux_dn_dir ({ncol_x*ncol_y, nlev}),
    flux_net    ({ncol_x*ncol_y, nlev}),
    flux_tod_dn ({ncol_x, ncol_y}),
    flux_tod_up ({ncol_x, ncol_y}),
    flux_sfc_dir({ncol_x, ncol_y}),
    flux_sfc_dif({ncol_x, ncol_y}),
    flux_sfc_up ({ncol_x, ncol_y}),
    flux_abs_dir({ncol_x, ncol_y, nlev-1}),
    flux_abs_dif({ncol_x, ncol_y, nlev-1})
{}


void Fluxes_broadband_rt::net_flux()
{
    const int ncol = this->flux_dn.dim(1);
    const int nlev = this->flux_dn.dim(2);

    const int block_lev = 16;
    const int block_col = 16;

    const int grid_col = ncol/block_col + (ncol%block_col > 0);
    const int grid_lev = nlev/block_lev + (nlev%block_lev > 0);

    dim3 grid_gpu(grid_col, grid_lev);
    dim3 block_gpu(block_col, block_lev);

    net_broadband_precalc<<<grid_gpu, block_gpu>>>(
            ncol, nlev, this->flux_dn.ptr(), this->flux_up.ptr(), this->flux_net.ptr());
}


void Fluxes_broadband_rt::reduce(
    const Array_gpu<Float,3>& gpt_flux_up, const Array_gpu<Float,3>& gpt_flux_dn,
    const std::unique_ptr<Optical_props_arry_rt>& spectral_disc,
    const Bool top_at_1)
{
    const int ncol = gpt_flux_up.dim(1);
    const int nlev = gpt_flux_up.dim(2);
    const int ngpt = gpt_flux_up.dim(3);

    const int block_lev = 16;
    const int block_col = 16;

    const int grid_col = ncol/block_col + (ncol%block_col > 0);
    const int grid_lev = nlev/block_lev + (nlev%block_lev > 0);

    dim3 grid_gpu(grid_col, grid_lev);
    dim3 block_gpu(block_col, block_lev);

    sum_broadband<<<grid_gpu, block_gpu>>>(
            ncol, nlev, ngpt, gpt_flux_up.ptr(), this->flux_up.ptr());

    sum_broadband<<<grid_gpu, block_gpu>>>(
            ncol, nlev, ngpt, gpt_flux_dn.ptr(), this->flux_dn.ptr());

    net_broadband_precalc<<<grid_gpu, block_gpu>>>(
            ncol, nlev, this->flux_dn.ptr(), this->flux_up.ptr(), this->flux_net.ptr());
}

//// CvH: unnecessary code duplication.

void Fluxes_broadband_rt::reduce(
    const Array_gpu<Float,3>& gpt_flux_up, const Array_gpu<Float,3>& gpt_flux_dn, const Array_gpu<Float,3>& gpt_flux_dn_dir,
    const std::unique_ptr<Optical_props_arry_rt>& spectral_disc,
    const Bool top_at_1)
{
    const int ncol = gpt_flux_up.dim(1);
    const int nlev = gpt_flux_up.dim(2);
    const int ngpt = gpt_flux_up.dim(3);

    reduce(gpt_flux_up, gpt_flux_dn, spectral_disc, top_at_1);

    const int block_lev = 16;
    const int block_col = 16;

    const int grid_col = ncol/block_col + (ncol%block_col > 0);
    const int grid_lev = nlev/block_lev + (nlev%block_lev > 0);

    dim3 grid_gpu(grid_col, grid_lev);
    dim3 block_gpu(block_col, block_lev);

    sum_broadband<<<grid_gpu, block_gpu>>>(
            ncol, nlev, ngpt, gpt_flux_dn_dir.ptr(), this->flux_dn_dir.ptr());
}


Fluxes_byband_rt::Fluxes_byband_rt(const int ncol_x, const int ncol_y, const int nlev, const int nbnd) :
    Fluxes_broadband_rt(ncol_x, ncol_y, nlev),
    bnd_flux_up    ({ncol_x * ncol_y, nlev, nbnd}),
    bnd_flux_dn    ({ncol_x * ncol_y, nlev, nbnd}),
    bnd_flux_dn_dir({ncol_x * ncol_y, nlev, nbnd}),
    bnd_flux_net   ({ncol_x * ncol_y, nlev, nbnd})
{}


void Fluxes_byband_rt::reduce(
    const Array_gpu<Float,3>& gpt_flux_up,
    const Array_gpu<Float,3>& gpt_flux_dn,
    const std::unique_ptr<Optical_props_arry_rt>& spectral_disc,
    const Bool top_at_1)
{
    const int ncol = gpt_flux_up.dim(1);
    const int nlev = gpt_flux_up.dim(2);
    const int ngpt = spectral_disc->get_ngpt();
    const int nbnd = spectral_disc->get_nband();

    const Array_gpu<int,2>& band_lims = spectral_disc->get_band_lims_gpoint();
    const int block_bnd = 1;
    const int block_lev = 16;
    const int block_col = 16;

    const int grid_col = ncol/block_col + (ncol%block_col > 0);
    const int grid_lev = nlev/block_lev + (nlev%block_lev > 0);
    const int grid_bnd = nbnd/block_bnd + (nbnd%block_bnd > 0);

    dim3 grid_gpu(grid_col, grid_lev, grid_bnd);
    dim3 block_gpu(block_col, block_lev, grid_bnd);

    Fluxes_broadband_rt::reduce(
            gpt_flux_up, gpt_flux_dn,
            spectral_disc, top_at_1);

    sum_byband<<<grid_gpu, block_gpu>>>(
            ncol, nlev, ngpt, nbnd, band_lims.ptr(),
            gpt_flux_up.ptr(), this->bnd_flux_up.ptr());

    sum_byband<<<grid_gpu, block_gpu>>>(
            ncol, nlev, ngpt, nbnd, band_lims.ptr(),
            gpt_flux_dn.ptr(), this->bnd_flux_dn.ptr());

    net_byband_full<<<grid_gpu, block_gpu>>>(
            ncol, nlev, ngpt, nbnd, band_lims.ptr(),
            this->bnd_flux_dn.ptr(), this->bnd_flux_up.ptr(), this->bnd_flux_net.ptr());
}

// CvH: a lot of code duplication.

void Fluxes_byband_rt::reduce(
    const Array_gpu<Float,3>& gpt_flux_up,
    const Array_gpu<Float,3>& gpt_flux_dn,
    const Array_gpu<Float,3>& gpt_flux_dn_dir,
    const std::unique_ptr<Optical_props_arry_rt>& spectral_disc,
    const Bool top_at_1)
{
    const int ncol = gpt_flux_up.dim(1);
    const int nlev = gpt_flux_up.dim(2);
    const int ngpt = spectral_disc->get_ngpt();
    const int nbnd = spectral_disc->get_nband();

    const Array_gpu<int,2>& band_lims = spectral_disc->get_band_lims_gpoint();

    Fluxes_broadband_rt::reduce(
            gpt_flux_up, gpt_flux_dn, gpt_flux_dn_dir,
            spectral_disc, top_at_1);

    reduce(gpt_flux_up, gpt_flux_dn, spectral_disc, top_at_1);

    const int block_bnd = 1;
    const int block_lev = 16;
    const int block_col = 16;

    const int grid_col = ncol/block_col + (ncol%block_col > 0);
    const int grid_lev = nlev/block_lev + (nlev%block_lev > 0);
    const int grid_bnd = nbnd/block_bnd + (nbnd%block_bnd > 0);

    dim3 grid_gpu(grid_col, grid_lev, grid_bnd);
    dim3 block_gpu(block_col, block_lev, grid_bnd);

    sum_byband<<<grid_gpu, block_gpu>>>(
            ncol, nlev, ngpt, nbnd, band_lims.ptr(),
            gpt_flux_dn_dir.ptr(), this->bnd_flux_dn_dir.ptr());
}


#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>

#include "Raytracer.h"
#include "Array.h"
#include "Optical_props_rt.h"

#include "raytracer_definitions.h"
#include "raytracer_functions.h"
#include "raytracer_kernels.h"

#include "gas_optics_rrtmgp_kernels_cuda_rt.h"


namespace
{
    using namespace Raytracer_functions;

    template<typename T>
    T* allocate_gpu(const int length)
    {
        T* data_ptr = Tools_gpu::allocate_gpu<T>(length);
        return data_ptr;
    }


    template<typename T>
    void copy_to_gpu(T* gpu_data, const T* cpu_data, const int length)
    {
        cuda_safe_call(hipMemcpy(gpu_data, cpu_data, length*sizeof(T), hipMemcpyHostToDevice));
    }


    template<typename T>
    void copy_from_gpu(T* cpu_data, const T* gpu_data, const int length)
    {
        cuda_safe_call(hipMemcpy(cpu_data, gpu_data, length*sizeof(T), hipMemcpyDeviceToHost));
    }


    __global__
    void create_knull_grid(
            const Vector<int> grid_cells, const Float k_ext_null_min,
            const Vector<int> kn_grid,
            const Float* __restrict__ k_ext, Float* __restrict__ k_null_grid)
    {
        const int grid_x = blockIdx.x*blockDim.x + threadIdx.x;
        const int grid_y = blockIdx.y*blockDim.y + threadIdx.y;
        const int grid_z = blockIdx.z*blockDim.z + threadIdx.z;
        if ( ( grid_x < kn_grid.x) && ( grid_y < kn_grid.y) && ( grid_z < kn_grid.z))
        {
            const Float fx = Float(grid_cells.x) / Float(kn_grid.x);
            const Float fy = Float(grid_cells.y) / Float(kn_grid.y);
            const Float fz = Float(grid_cells.z) / Float(kn_grid.z);

            const int x0 = grid_x*fx;
            const int x1 = floor((grid_x+1)*fx);
            const int y0 = grid_y*fy;
            const int y1 = floor((grid_y+1)*fy);
            const int z0 = grid_z*fz;
            const int z1 = floor((grid_z+1)*fz);

            const int ijk_grid = grid_x + grid_y*kn_grid.x + grid_z*kn_grid.y*kn_grid.x;
            Float k_null = k_ext_null_min;

            for (int k=z0; k<z1; ++k)
                for (int j=y0; j<y1; ++j)
                    for (int i=x0; i<x1; ++i)
                    {
                        const int ijk_in = i + j*grid_cells.x + k*grid_cells.x*grid_cells.y;
                        k_null = max(k_null, k_ext[ijk_in]);
                    }

            k_null_grid[ijk_grid] = k_null;
        }
    }


    __global__
    void bundles_optical_props(
            const Vector<int> grid_cells, const Vector<Float> grid_d,
            const Float* __restrict__ tau_tot, const Float* __restrict__ ssa_tot,
            const Float* __restrict__ tau_cld, const Float* __restrict__ ssa_cld, const Float* __restrict__ asy_cld,
            const Float* __restrict__ tau_aer, const Float* __restrict__ ssa_aer, const Float* __restrict__ asy_aer,
            Float* __restrict__ k_ext, Optics_scat* __restrict__ scat_asy)
    {
        const int icol_x = blockIdx.x*blockDim.x + threadIdx.x;
        const int icol_y = blockIdx.y*blockDim.y + threadIdx.y;
        const int iz = blockIdx.z*blockDim.z + threadIdx.z;

        if ( (icol_x < grid_cells.x) && (icol_y < grid_cells.y) && (iz < grid_cells.z) )
        {
            const int idx = icol_x + icol_y*grid_cells.x + iz*grid_cells.y*grid_cells.x;
            const Float kext_tot = tau_tot[idx] / grid_d.z;
            const Float kext_cld = tau_cld[idx] / grid_d.z;
            const Float kext_aer = tau_aer[idx] / grid_d.z;
            const Float ksca_cld = kext_cld * ssa_cld[idx];
            const Float ksca_aer = kext_aer * ssa_aer[idx];
            const Float ksca_gas = kext_tot * ssa_tot[idx] - ksca_cld - ksca_aer;
            k_ext[idx] = tau_tot[idx] / grid_d.z;

            scat_asy[idx].k_sca_gas = ksca_gas;
            scat_asy[idx].k_sca_cld = ksca_cld;
            scat_asy[idx].k_sca_aer = ksca_aer;
            scat_asy[idx].asy_cld = asy_cld[idx];
            scat_asy[idx].asy_aer = asy_aer[idx];
        }
    }


    __global__
    void count_to_flux_2d(
            const Vector<int> grid_cells, const Float photons_per_col, const Float toa_src,
            const Float* __restrict__ count_1, const Float* __restrict__ count_2, const Float* __restrict__ count_3, const Float* __restrict__ count_4, const Float* __restrict__ count_5,
            Float* __restrict__ flux_1, Float* __restrict__ flux_2, Float* __restrict__ flux_3, Float* __restrict__ flux_4, Float* __restrict__ flux_5)
    {
        const int icol_x = blockIdx.x*blockDim.x + threadIdx.x;
        const int icol_y = blockIdx.y*blockDim.y + threadIdx.y;

        if ( ( icol_x < grid_cells.x) && ( icol_y < grid_cells.y) )
        {
            const int idx = icol_x + icol_y*grid_cells.x;
            const Float flux_per_ray = toa_src / photons_per_col;
            flux_1[idx] = count_1[idx] * flux_per_ray;
            flux_2[idx] = count_2[idx] * flux_per_ray;
            flux_3[idx] = count_3[idx] * flux_per_ray;
            flux_4[idx] = count_4[idx] * flux_per_ray;
            flux_5[idx] = count_5[idx] * flux_per_ray;
        }
    }

    __global__
    void count_to_flux_3d(
            const Vector<int> grid_cells, const Float photons_per_col,
            const Vector<Float> grid_d, const Float toa_src,
            const Float* __restrict__ count_1, const Float* __restrict__ count_2,
            Float* __restrict__ flux_1, Float* __restrict__ flux_2)
    {
        const int icol_x = blockIdx.x*blockDim.x + threadIdx.x;
        const int icol_y = blockIdx.y*blockDim.y + threadIdx.y;
        const int iz = blockIdx.z*blockDim.z + threadIdx.z;

        if ( ( icol_x < grid_cells.x) && ( icol_y < grid_cells.y) && ( iz < grid_cells.z))
        {
            const int idx = icol_x + icol_y*grid_cells.x + iz*grid_cells.x*grid_cells.y;
            const Float flux_per_ray = toa_src / photons_per_col;
            flux_1[idx] = count_1[idx] * flux_per_ray / grid_d.z;
            flux_2[idx] = count_2[idx] * flux_per_ray / grid_d.z;
        }
    }
}


Raytracer::Raytracer()
{
    hiprandDirectionVectors32_t* qrng_vectors;
    hiprandGetDirectionVectors32(
                &qrng_vectors,
                HIPRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6);
    unsigned int* qrng_constants;
    hiprandGetScrambleConstants32(&qrng_constants);

    this->qrng_vectors_gpu = allocate_gpu<hiprandDirectionVectors32_t>(2);
    this->qrng_constants_gpu = allocate_gpu<unsigned int>(2);

    copy_to_gpu(qrng_vectors_gpu, qrng_vectors, 2);
    copy_to_gpu(qrng_constants_gpu, qrng_constants, 2);
}


void Raytracer::trace_rays(
        const int igpt,
        const Int photons_per_pixel,
        const Vector<int> grid_cells,
        const Vector<Float> grid_d,
        const Vector<int> kn_grid,
        const Array_gpu<Float,2>& mie_cdf,
        const Array_gpu<Float,3>& mie_ang,
        const Array_gpu<Float,2>& tau_total,
        const Array_gpu<Float,2>& ssa_total,
        const Array_gpu<Float,2>& tau_cloud,
        const Array_gpu<Float,2>& ssa_cloud,
        const Array_gpu<Float,2>& asy_cloud,
        const Array_gpu<Float,2>& tau_aeros,
        const Array_gpu<Float,2>& ssa_aeros,
        const Array_gpu<Float,2>& asy_aeros,
        const Array_gpu<Float,2>& r_eff,
        const Array_gpu<Float,2>& surface_albedo,
        const Float zenith_angle,
        const Float azimuth_angle,
        const Float tod_inc_direct,
        const Float tod_inc_diffuse,
        Array_gpu<Float,2>& flux_tod_dn,
        Array_gpu<Float,2>& flux_tod_up,
        Array_gpu<Float,2>& flux_sfc_dir,
        Array_gpu<Float,2>& flux_sfc_dif,
        Array_gpu<Float,2>& flux_sfc_up,
        Array_gpu<Float,3>& flux_abs_dir,
        Array_gpu<Float,3>& flux_abs_dif)
{
    // set of block and grid dimensions used in data processing kernels - requires some proper tuning later
    const int block_col_x = 8;
    const int block_col_y = 8;
    const int block_z = 4;

    const int grid_col_x = grid_cells.x/block_col_x + (grid_cells.x%block_col_x > 0);
    const int grid_col_y = grid_cells.y/block_col_y + (grid_cells.y%block_col_y > 0);
    const int grid_z = grid_cells.z/block_z + (grid_cells.z%block_z > 0);

    dim3 grid_2d(grid_col_x, grid_col_y);
    dim3 block_2d(block_col_x, block_col_y);
    dim3 grid_3d(grid_col_x, grid_col_y, grid_z);
    dim3 block_3d(block_col_x, block_col_y, block_z);

    // Bundle optical properties in struct
    Array_gpu<Float,3> k_ext({grid_cells.x, grid_cells.y, grid_cells.z});
    Array_gpu<Optics_scat,3> scat_asy({grid_cells.x, grid_cells.y, grid_cells.z});

    bundles_optical_props<<<grid_3d, block_3d>>>(
            grid_cells, grid_d,
            tau_total.ptr(), ssa_total.ptr(),
            tau_cloud.ptr(), ssa_cloud.ptr(), asy_cloud.ptr(),
            tau_aeros.ptr(), ssa_aeros.ptr(), asy_aeros.ptr(),
            k_ext.ptr(), scat_asy.ptr());

    // create k_null_grid
    const int block_kn_x = 8;
    const int block_kn_y = 8;
    const int block_kn_z = 4;

    const int grid_kn_x = kn_grid.x/block_kn_x + (kn_grid.x%block_kn_x > 0);
    const int grid_kn_y = kn_grid.y/block_kn_y + (kn_grid.y%block_kn_y > 0);
    const int grid_kn_z = kn_grid.z/block_kn_z + (kn_grid.z%block_kn_z > 0);

    dim3 grid_kn(grid_kn_x, grid_kn_y, grid_kn_z);
    dim3 block_kn(block_kn_x, block_kn_y, block_kn_z);

    Array_gpu<Float,3> k_null_grid({kn_grid.x, kn_grid.y, kn_grid.z});
    const Float k_ext_null_min = Float(1e-3);

    create_knull_grid<<<grid_kn, block_kn>>>(
            grid_cells, k_ext_null_min,
            kn_grid,
            k_ext.ptr(), k_null_grid.ptr());

    // initialise output arrays and set to 0
    Array_gpu<Float,2> tod_dn_count({grid_cells.x, grid_cells.y});
    Array_gpu<Float,2> tod_up_count({grid_cells.x, grid_cells.y});
    Array_gpu<Float,2> surface_down_direct_count({grid_cells.x, grid_cells.y});
    Array_gpu<Float,2> surface_down_diffuse_count({grid_cells.x, grid_cells.y});
    Array_gpu<Float,2> surface_up_count({grid_cells.x, grid_cells.y});
    Array_gpu<Float,3> atmos_direct_count({grid_cells.x, grid_cells.y, grid_cells.z});
    Array_gpu<Float,3> atmos_diffuse_count({grid_cells.x, grid_cells.y, grid_cells.z});

    Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(grid_cells.x, grid_cells.y, tod_dn_count.ptr());
    Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(grid_cells.x, grid_cells.y, tod_up_count.ptr());
    Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(grid_cells.x, grid_cells.y, surface_down_direct_count.ptr());
    Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(grid_cells.x, grid_cells.y, surface_down_diffuse_count.ptr());
    Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(grid_cells.x, grid_cells.y, surface_up_count.ptr());
    Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(grid_cells.x, grid_cells.y, grid_cells.z, atmos_direct_count.ptr());
    Gas_optics_rrtmgp_kernels_cuda_rt::zero_array(grid_cells.x, grid_cells.y, grid_cells.z, atmos_diffuse_count.ptr());

    // domain sizes
    const Vector<Float> grid_size = grid_d * grid_cells;

    // direction of direct rays. Take into account that azimuth is 0 north and increases clockwise
    const Vector<Float> sun_direction = {-std::sin(zenith_angle) * std::cos(Float(0.5*M_PI) - azimuth_angle),
                                         -std::sin(zenith_angle) * std::sin(Float(0.5*M_PI) - azimuth_angle),
                                         -std::cos(zenith_angle)};

    // smallest two power that is larger than grid dimension (minimum of 2 is currently required)
    const Int qrng_grid_x = std::max(Float(2), pow(Float(2.), ceil(std::log2(Float(grid_cells.x)))) );
    const Int qrng_grid_y = std::max(Float(2), pow(Float(2.), ceil(std::log2(Float(grid_cells.y)))) );
   
    // total number of photons
    const Int photons_total = photons_per_pixel * qrng_grid_x * qrng_grid_y;

    // number of photons per thread, this should a power of 2 and nonzero
    Float photons_per_thread_tmp = std::max(Float(1), static_cast<Float>(photons_total) / (rt_kernel_grid * rt_kernel_block));
    Int photons_per_thread = pow(Float(2.), std::floor(std::log2(photons_per_thread_tmp)));
    
    dim3 grid,block;
    
    // with very low number of columns and photons_per_pixel, we may have too many threads firing a single photons, actually exceeding photons_per pixel
    // In that case, reduce grid and block size
    Int actual_photons_per_pixel = photons_per_thread * rt_kernel_grid * rt_kernel_block / (qrng_grid_x * qrng_grid_y);
    
    int rt_kernel_grid_size = rt_kernel_grid;
    int rt_kernel_block_size = rt_kernel_block;
    while ( (actual_photons_per_pixel > photons_per_pixel) )
    {
        if (rt_kernel_grid_size > 1)
            rt_kernel_grid_size /= 2;
        else
            rt_kernel_block_size /= 2;        
        
        photons_per_thread_tmp = std::max(Float(1), static_cast<Float>(photons_total) / (rt_kernel_grid_size * rt_kernel_block_size));
        photons_per_thread = pow(Float(2.), std::floor(std::log2(photons_per_thread_tmp)));
        actual_photons_per_pixel = photons_per_thread * rt_kernel_grid_size * rt_kernel_block_size / (qrng_grid_x * qrng_grid_y);
    }
    
    grid = {rt_kernel_grid_size};
    block = {rt_kernel_block_size};
    
    const int mie_table_size = mie_cdf.size();
    
    const int qrng_gpt_offset = (igpt-1) * rt_kernel_grid_size * rt_kernel_block_size * photons_per_thread;
    ray_tracer_kernel<<<grid, block,sizeof(Float)*mie_table_size>>>(
            photons_per_thread,
            qrng_grid_x,
            qrng_grid_y,
            qrng_gpt_offset,
            k_null_grid.ptr(),
            tod_dn_count.ptr(),
            tod_up_count.ptr(),
            surface_down_direct_count.ptr(),
            surface_down_diffuse_count.ptr(),
            surface_up_count.ptr(),
            atmos_direct_count.ptr(),
            atmos_diffuse_count.ptr(),
            k_ext.ptr(), scat_asy.ptr(),
            r_eff.ptr(),
            tod_inc_direct,
            tod_inc_diffuse,
            surface_albedo.ptr(),
            grid_size, grid_d, grid_cells, kn_grid,
            sun_direction,
            this->qrng_vectors_gpu, this->qrng_constants_gpu,
            mie_cdf.ptr(), mie_ang.ptr(), mie_table_size);

    // convert counts to fluxes

    const Float toa_src = tod_inc_direct + tod_inc_diffuse;
    count_to_flux_2d<<<grid_2d, block_2d>>>(
            grid_cells, photons_per_pixel,
            toa_src,
            tod_dn_count.ptr(),
            tod_up_count.ptr(),
            surface_down_direct_count.ptr(),
            surface_down_diffuse_count.ptr(),
            surface_up_count.ptr(),
            flux_tod_dn.ptr(),
            flux_tod_up.ptr(),
            flux_sfc_dir.ptr(),
            flux_sfc_dif.ptr(),
            flux_sfc_up.ptr());

    count_to_flux_3d<<<grid_3d, block_3d>>>(
            grid_cells, photons_per_pixel,
            grid_d,
            toa_src,
            atmos_direct_count.ptr(),
            atmos_diffuse_count.ptr(),
            flux_abs_dir.ptr(),
            flux_abs_dif.ptr());





}

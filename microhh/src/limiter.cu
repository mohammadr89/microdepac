#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2023 Chiel van Heerwaarden
 * Copyright (c) 2011-2023 Thijs Heus
 * Copyright (c) 2014-2023 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <algorithm>
#include <iostream>
#include "master.h"
#include "grid.h"
#include "fields.h"
#include "stats.h"
#include "limiter.h"
#include "tools.h"
#include "constants.h"

namespace
{
    template<typename TF>__global__
    void tendency_limiter(
            TF* const __restrict__ at,
            const TF* const __restrict__ a,
            const TF min_value,
            const TF dt, const TF dti,
            const int istart, const int iend,
            const int jstart, const int jend,
            const int kstart, const int kend,
            const int jj, const int kk)
    {
        const int i  = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j  = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k  = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;

            const TF a_new = a[ijk] + dt*at[ijk];
            at[ijk] += (a_new < min_value) ? (-a_new + min_value) * dti : TF(0.);
        }
    }
}

#ifdef USECUDA
template <typename TF>
void Limiter<TF>::exec(double dt, Stats<TF>& stats)
{
    const Grid_data<TF>& gd = grid.get_grid_data();
    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kmax);
    dim3 blockGPU(blocki, blockj, 1);

    const TF dti = 1./dt;

    // Add epsilon, to make sure the final result ends just above zero.
    // NOTE: don't use `eps<TF>` here; `eps<float>` is too large
    //       as a lower limit for e.g. hydrometeors or chemical species.
    constexpr TF min_value = std::numeric_limits<double>::epsilon();

    for (auto& name : limit_list)
    {
        tendency_limiter<TF><<<gridGPU, blockGPU>>>(
            fields.at.at(name)->fld_g,
            fields.ap.at(name)->fld_g,
            min_value,
            dt, dti,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.kstart, gd.kend,
            gd.icells, gd.ijcells);
        cuda_check_error();

        stats.calc_tend(*fields.at.at(name), tend_name);
    }

    if (limit_sgstke)
    {
        tendency_limiter<TF><<<gridGPU, blockGPU>>>(
            fields.at.at("sgstke")->fld_g,
            fields.ap.at("sgstke")->fld_g,
            Constants::sgstke_min<TF>,
            dt, dti,
            gd.istart, gd.iend,
            gd.jstart, gd.jend,
            gd.kstart, gd.kend,
            gd.icells, gd.ijcells);
        cuda_check_error();

        stats.calc_tend(*fields.at.at("sgstke"), tend_name);
    }
}
#endif


#ifdef FLOAT_SINGLE
template class Limiter<float>;
#else
template class Limiter<double>;
#endif
